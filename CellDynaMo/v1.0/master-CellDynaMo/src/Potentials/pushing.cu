#include "hip/hip_runtime.h"
/*
 *pushing module
 *
 *  Created on: Mar 15, 2018
 *	Edited on:	Mar 30, 2019
 *      Author: kliuchnikov
 */

#include "pushing.cuh"		

__global__ void pushing(float3* r, float3* f, int N, int* type, int kn, long long int step, int* n_force, float* f_force, Param* d_parameters){
	int i, j, h;
    int N_pol = d_parameters->pole_n;
    int N_mt = d_parameters->mt_n;
    int N_mt_max = d_parameters->max_mt_n;
    int N_kt = d_parameters->kin_n;
    float mt_rad = d_parameters->mt_r;
    float kt_rad = d_parameters->kt_r;
    float e_lj = d_parameters->e_rep/0.23;
    float zone = d_parameters->zone;
    float big_r = d_parameters->big_r;
    int chrom_cond = d_parameters->chrom;
    int x_num = d_parameters->x_num;
    int y_num = d_parameters->y_num;
    int chrom_bn = chrom_cond*d_parameters->chrom_num;
	i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < N){
		if (type[i] == PLUS_DET || type[i] == MT_REG || type[i] == LEFT_MINUS || type[i] == RIGHT_MINUS || type[i] == PLUS_ATT || type[i] == PLUS_DET_INVALID){
			float3 ri = r[i];
		    for (h = 0; h < N_kt; h++){
		        j = N_pol*N_mt*N_mt_max + (h + 1)*kn - 1 - chrom_bn;

		        float3 rj = r[j];

		        float dx = rj.x - ri.x;
		        float dy = rj.y - ri.y;
		        float dz = rj.z - ri.z;

		        float dr = sqrtf(dx*dx + dy*dy + dz*dz);

                float high = big_r - kt_rad + zone;
			    float limit = sqrtf(pow(zone, 2) + pow(big_r, 2) - pow(high, 2));
                if (dr < limit + 4*mt_rad){
                    float3 kt1;
				    float3 kt2;
                    if (h % 2 == 0){
					    kt1 = r[N_pol*N_mt*N_mt_max + (h + 1)*kn - 1 - chrom_bn];
					    kt2 = r[N_pol*N_mt*N_mt_max + (h + 2)*kn - 1 - chrom_bn];
				    }
				    else	{
					    kt1 = r[N_pol*N_mt*N_mt_max + h*kn - 1 - chrom_bn];
					    kt2 = r[N_pol*N_mt*N_mt_max + (h + 1)*kn - 1 - chrom_bn];
				    }
                    float3 normal;
			        normal.x = kt2.x - kt1.x;
			        normal.y = kt2.y - kt1.y;
			        normal.z = kt2.z - kt1.z; 
			        float dn = sqrt(normal.x*normal.x + normal.y*normal.y + normal.z*normal.z);
                    float3 delta;
			        delta.x = zone*normal.x/dn;
			        delta.y = zone*normal.y/dn;
			        delta.z = zone*normal.z/dn;

			        float3 right, left;
			        right.x = kt2.x + delta.x;
			        right.y = kt2.y + delta.y;
			        right.z = kt2.z + delta.z;

			        left.x = kt1.x - delta.x;
			        left.y = kt1.y - delta.y;
			        left.z = kt1.z - delta.z;
                    
                    if ((normal.x*(ri.x - right.x) + normal.y*(ri.y - right.y) + normal.z*(ri.z - right.z)) < 0 && (normal.x*(ri.x - left.x) + normal.y*(ri.y - left.y) + normal.z*(ri.z - left.z)) > 0){
                        if (dr > limit - mt_rad){
                            float df = -6*e_lj*powf(limit/dr, 12)*powf(dr, -1)*powf(dr, -1);
                            atomicAdd(&f[i].x, df*dx);
	                        atomicAdd(&f[i].y, df*dy);  		
	                        atomicAdd(&f[i].z, df*dz);

	                        atomicAdd(&f[j].x, -df*dx);
	                        atomicAdd(&f[j].y, -df*dy);		
	                        atomicAdd(&f[j].z, -df*dz);

                             if (type[j] == LEFT_KT  && step == 0){
                                atomicAdd(&n_force[h*5], 1);
                                atomicAdd(&f_force[h*5 + 1], -df*1.66);
                                atomicAdd(&n_force[h*5 + 2], 1); 
                            }
                            else    if (type[j] == RIGHT_KT  && step == 0){
                                atomicAdd(&n_force[h*5], 1);
                                atomicAdd(&f_force[h*5 + 1], -df*1.66);
                                atomicAdd(&n_force[h*5 + 2], 1); 
                            }
                        }       
                    }
                    else    {
                        float minR = 2*mt_rad;
                        float maxR = kt_rad;
                        for (int ks = N_pol*N_mt*N_mt_max + h*kn; ks < N_pol*N_mt*N_mt_max + (h + 1)*kn - 1 - chrom_bn; ks++){
                            float3 rk = r[ks];
                            float dxk = rk.x - ri.x;
		                    float dyk = rk.y - ri.y;
		                    float dzk = rk.z - ri.z;

		                    float drk = sqrtf(dxk*dxk + dyk*dyk + dzk*dzk);
                            if (drk < minR){
                                float dxc = rk.x - rj.x;
		                        float dyc = rk.y - rj.y;
		                        float dzc = rk.z - rj.z;

		                        float drc = sqrtf(dxc*dxc + dyc*dyc + dzc*dzc);
                                minR = drk;
                                maxR = drc;
                            }
                        } 
                        if (minR < 2*mt_rad && dr > maxR - mt_rad){
                    		float df = -6*e_lj*powf(maxR/dr, 12)*powf(dr, -1)*powf(dr, -1);                            
                            atomicAdd(&f[i].x, df*dx);
	                        atomicAdd(&f[i].y, df*dy);  		
	                        atomicAdd(&f[i].z, df*dz);

	                        atomicAdd(&f[j].x, -df*dx);
	                        atomicAdd(&f[j].y, -df*dy);		
	                        atomicAdd(&f[j].z, -df*dz);

                            if (type[j] == LEFT_KT  && step == 0){
                                atomicAdd(&n_force[h*5], 1);
                                atomicAdd(&f_force[h*5 + 1], -df*1.66);
                                atomicAdd(&n_force[h*5 + 2], 1); 
                            }
                            else    if (type[j] == RIGHT_KT  && step == 0){
                                atomicAdd(&n_force[h*5], 1);
                                atomicAdd(&f_force[h*5 + 1], -df*1.66);
                                atomicAdd(&n_force[h*5 + 2], 1); 
                            }
                        }                  
                    }
                }
		    }
            for (int p = 0; p < N_kt; p++){
                for (int ch = 1; ch <= chrom_bn; ch++){
                    int k = N_pol*N_mt*N_mt_max + (p + 1)*kn - ch;

                    float3 rj = r[k];

		            float dx = rj.x - ri.x;
		            float dy = rj.y - ri.y;
		            float dz = rj.z - ri.z;

		            float dr = sqrtf(dx*dx + dy*dy + dz*dz);
                    if (dr < kt_rad + 2*mt_rad && dr > kt_rad - mt_rad){
                        float df = -6*e_lj*powf((kt_rad + 3*mt_rad)/dr, 12)*powf(dr, -1)*powf(dr, -1)/10;
                        //printf("%f\t%f\t%f\t%f\n", e_lj, dr, kt_rad + 3*mt_rad/dr, df);
                        atomicAdd(&f[i].x, df*dx);
                        atomicAdd(&f[i].y, df*dy);  		
                        atomicAdd(&f[i].z, df*dz);

                        atomicAdd(&f[k].x, -df*dx);
                        atomicAdd(&f[k].y, -df*dy);		
                        atomicAdd(&f[k].z, -df*dz);
                        if (type[k - 5 + ch] == LEFT_KT/*  && step == 0*/){
                            atomicAdd(&n_force[p*5], 1);
                            atomicAdd(&f_force[p*5 + 1], -df*1.66);
                            atomicAdd(&n_force[p*5 + 2], 1);   
                                 
                            /*atomicAdd(&n_force[p*5 + 1], 1);
                            atomicAdd(&f_force[p*5 + 1], -df*1.66);
                            atomicAdd(&n_force[p*5 + 2], 1);*/
                        }
                        else    if (type[k - 5 + ch] == RIGHT_KT/*  && step == 0*/){
                            atomicAdd(&n_force[p*5], 1);
                            atomicAdd(&f_force[p*5 + 1], -df*1.66);
                            atomicAdd(&n_force[p*5 + 2], 1);

                            /*atomicAdd(&n_force[p*5], 1);
                            atomicAdd(&f_force[p*5], -df*1.66);
                            atomicAdd(&n_force[p*5 + 3], 1);*/
                        }
                    }
                }
                for (int sh = x_num*y_num; sh < kn; sh++){
                    int k = N_pol*N_mt*N_mt_max + p*kn + sh;
    
                    float3 rj = r[k];

		            float dx = rj.x - ri.x;
		            float dy = rj.y - ri.y;
		            float dz = rj.z - ri.z;

		            float dr = sqrtf(dx*dx + dy*dy + dz*dz);
                    if (dr < 80.0){
                        float3 kt1;
				        float3 kt2;
                        if (p % 2 == 0){
					        kt1 = r[N_pol*N_mt*N_mt_max + (p + 1)*kn - 1 - chrom_bn];
					        kt2 = r[N_pol*N_mt*N_mt_max + (p + 2)*kn - 1 - chrom_bn];
				        
	                        float dx1 = kt1.x - ri.x;
		                    float dy1 = kt1.y - ri.y;
		                    float dz1 = kt1.z - ri.z;

	                        float dx2 = kt2.x - ri.x;
		                    float dy2 = kt2.y - ri.y;
		                    float dz2 = kt2.z - ri.z;
                            
		                    float dr1 = sqrtf(dx1*dx1 + dy1*dy1 + dz1*dz1);
		                    float dr2 = sqrtf(dx2*dx2 + dy2*dy2 + dz2*dz2);

                            if (dr1 < dr2){
                                float df = -6*e_lj*powf(80.0/dr, 12)*powf(dr, -1)*powf(dr, -1)/10;
                                //printf("%f\n, df");
                                if (df > 100.0){
                                    df = 100.0;
                                } 
                                if (df < -100.0){
                                    df = -100.0;
                                }                                                               
                                //printf("%f\t%f\t%f\t%f\n", e_lj, dr, kt_rad + 3*mt_rad/dr, df);
                                atomicAdd(&f[i].x, df*dx);
                                atomicAdd(&f[i].y, df*dy);  		
                                atomicAdd(&f[i].z, df*dz);

                                atomicAdd(&f[N_pol*N_mt*N_mt_max + (p + 1)*kn - 1 - chrom_bn].x, -df*dx);
                                atomicAdd(&f[N_pol*N_mt*N_mt_max + (p + 1)*kn - 1 - chrom_bn].y, -df*dy);		
                                atomicAdd(&f[N_pol*N_mt*N_mt_max + (p + 1)*kn - 1 - chrom_bn].z, -df*dz);
                                
                                atomicAdd(&n_force[p*5], 1);
                                atomicAdd(&f_force[p*5 + 1], -df*1.66);
                                atomicAdd(&n_force[p*5 + 2], 1);   
                            }
                            if (dr2 < dr1){
                                float df = -6*e_lj*powf(80.0/dr, 12)*powf(dr, -1)*powf(dr, -1)/10;
                                if (df > 100.0){
                                    df = 100.0;
                                }
                                if (df < -100.0){
                                    df = -100.0;
                                } 
                                //printf("%f\t%f\t%f\t%f\n", e_lj, dr, kt_rad + 3*mt_rad/dr, df);
                                atomicAdd(&f[i].x, df*dx);
                                atomicAdd(&f[i].y, df*dy);  		
                                atomicAdd(&f[i].z, df*dz);

                                atomicAdd(&f[N_pol*N_mt*N_mt_max + (p + 2)*kn - 1 - chrom_bn].x, -df*dx);
                                atomicAdd(&f[N_pol*N_mt*N_mt_max + (p + 2)*kn - 1 - chrom_bn].y, -df*dy);		
                                atomicAdd(&f[N_pol*N_mt*N_mt_max + (p + 2)*kn - 1 - chrom_bn].z, -df*dz);
                              
                                atomicAdd(&n_force[p*5], 1);
                                atomicAdd(&f_force[p*5 + 1], -df*1.66);
                                atomicAdd(&n_force[p*5 + 2], 1);
                            }
                        }
                    }
                }
            }
        }
    	//__syncthreads();
	}
}		
