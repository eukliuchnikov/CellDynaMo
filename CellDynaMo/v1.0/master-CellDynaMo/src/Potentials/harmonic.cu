#include "hip/hip_runtime.h"
#include "harmonic.cuh"		

__global__ void computeHarmonic(float3* r, float3* f, int N, int* type, int kn, float rk, int* harmonicKinCount, int* harmonicKin, float* harmonicKinRadii, float* kt_radius, Param* d_parameters, float* length){
    int N_pol = d_parameters->pole_n;
    int N_mt = d_parameters->mt_n;
    int N_kt = d_parameters->kin_n;
    int N_mt_max = d_parameters->max_mt_n;
    float K_kt = d_parameters->k_kt;
    float K_mt = d_parameters->k_mt;
    int MpK = d_parameters->mHkPm;
    int chrom_cond = d_parameters->chrom;
    int chrom_bn = chrom_cond*d_parameters->chrom_num;
    int x_num = d_parameters->x_num;
    int y_num = d_parameters->y_num;
    float kt_rad = d_parameters->kt_r;
	int i;	
	i = blockIdx.x*blockDim.x + threadIdx.x;	
	if (i < N){
		if (type[i] == MT_REG || type[i] == PLUS_DET || type[i] == PLUS_ATT || type[i] == PLUS_DET_INVALID){
            float r0 = length[i]/2.0;
   			float3 ri = r[i - 1];
			float3 rj = r[i];

			float dx = rj.x - ri.x;
			float dy = rj.y - ri.y;
			float dz = rj.z - ri.z;
	        
			float dr = sqrtf(dx*dx + dy*dy + dz*dz);
            float df = K_mt*(dr - r0)*powf(dr, -1);

		    atomicAdd(&f[i - 1].x, df*dx);
		    atomicAdd(&f[i - 1].y, df*dy);		
		    atomicAdd(&f[i - 1].z, df*dz);
            
			atomicAdd(&f[i].x, -df*dx);
			atomicAdd(&f[i].y, -df*dy);		
			atomicAdd(&f[i].z, -df*dz);
		}
        /*if (type[i] == LEFT_MINUS){
            float r0 = length[i]/2.0;
   			float3 ri = r[N - 2];
			float3 rj = r[i];

			float dx = rj.x - ri.x;
			float dy = rj.y - ri.y;
			float dz = rj.z - ri.z;
	    
			float dr = sqrtf(dx*dx + dy*dy + dz*dz);
            float df = K_mt*(dr - r0)*powf(dr, -1);

		    atomicAdd(&f[N - 2].x, df*dx);
		    atomicAdd(&f[N - 2].y, df*dy);		
		    atomicAdd(&f[N - 2].z, df*dz);
            
			atomicAdd(&f[i].x, -df*dx);
			atomicAdd(&f[i].y, -df*dy);		
			atomicAdd(&f[i].z, -df*dz);
		}
        if (type[i] == RIGHT_MINUS){
            float r0 = length[i]/2.0;
   			float3 ri = r[N - 1];
			float3 rj = r[i];

			float dx = rj.x - ri.x;
			float dy = rj.y - ri.y;
			float dz = rj.z - ri.z;
	    
			float dr = sqrtf(dx*dx + dy*dy + dz*dz);
            float df = K_mt*(dr - r0)*powf(dr, -1);

		    atomicAdd(&f[N - 1].x, df*dx);
		    atomicAdd(&f[N - 1].y, df*dy);		
		    atomicAdd(&f[N - 1].z, df*dz);
            
			atomicAdd(&f[i].x, -df*dx);
			atomicAdd(&f[i].y, -df*dy);		
			atomicAdd(&f[i].z, -df*dz);
		}*/
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//**************************************************************************************************************************************************************//
//between centres of kinetochores
//**************************************************************************************************************************************************************//
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		if (type[i] == LEFT_KT){
			float3 ri = r[i];
			int j = i + kn;
			float3 rj = r[j];

			float dx = rj.x - ri.x;
			float dy = rj.y - ri.y;
			float dz = rj.z - ri.z;

			float dr = sqrtf(dx*dx + dy*dy + dz*dz);
			float df = 0.5*K_kt*(dr - rk)*powf(dr, -1);

			atomicAdd(&f[i].x, df*dx);
			atomicAdd(&f[i].y, df*dy);		
			atomicAdd(&f[i].z, df*dz);

			atomicAdd(&f[j].x, -df*dx);
			atomicAdd(&f[j].y, -df*dy);		
			atomicAdd(&f[j].z, -df*dz);
            
            if (chrom_bn != 0){
                for (int chrom = 1; chrom < chrom_bn + 1; chrom++){
                    float3 rch1 = r[i + chrom];
			        int c1 = i + kn + chrom;
			        float3 rch2 = r[c1];

			        float dx1 = rch2.x - rch1.x;
			        float dy1 = rch2.y - rch1.y;
			        float dz1 = rch2.z - rch1.z;

			        float dr1 = sqrtf(dx1*dx1 + dy1*dy1 + dz1*dz1);
			        float df1 = 0.001*K_kt*(dr1 - rk)*powf(dr1, -1);
                    

                    atomicAdd(&f[i + chrom].x, df1*dx1);
			        atomicAdd(&f[i + chrom].y, df1*dy1);		
			        atomicAdd(&f[i + chrom].z, df1*dz1);

			        atomicAdd(&f[c1].x, -df1*dx1);
			        atomicAdd(&f[c1].y, -df1*dy1);		
			        atomicAdd(&f[c1].z, -df1*dz1);
                }

                for (int chrom = 1; chrom < chrom_bn; chrom += chrom_bn/2){
                    int k = i + chrom;
			        float3 rk = r[k];

			        float dxk = rk.x - ri.x;
			        float dyk = rk.y - ri.y;
			        float dzk = rk.z - ri.z;

			        float drk = sqrtf(dxk*dxk + dyk*dyk + dzk*dzk);
			        float dfk = 200*K_kt*(drk - kt_radius[k - N_pol*N_mt*N_mt_max])*powf(drk, -1);


			        atomicAdd(&f[i].x, dfk*dxk);
			        atomicAdd(&f[i].y, dfk*dyk);		
			        atomicAdd(&f[i].z, dfk*dzk);

			        atomicAdd(&f[k].x, -dfk*dxk);
			        atomicAdd(&f[k].y, -dfk*dyk);		
			        atomicAdd(&f[k].z, -dfk*dzk);
                }
            }
		}
        if (type[i] == RIGHT_KT && chrom_bn != 0){
            float3 ri = r[i];
            for (int chrom = 1; chrom < chrom_bn; chrom += chrom_bn/2){
                int k = i + chrom;
			    float3 rk = r[k];

			    float dxk = rk.x - ri.x;
			    float dyk = rk.y - ri.y;
			    float dzk = rk.z - ri.z;

			    float drk = sqrtf(dxk*dxk + dyk*dyk + dzk*dzk);
			    float dfk = 200*K_kt*(drk - kt_radius[k - N_pol*N_mt*N_mt_max])*powf(drk, -1);

			    atomicAdd(&f[i].x, dfk*dxk);
			    atomicAdd(&f[i].y, dfk*dyk);		
			    atomicAdd(&f[i].z, dfk*dzk);

			    atomicAdd(&f[k].x, -dfk*dxk);
			    atomicAdd(&f[k].y, -dfk*dyk);		
			    atomicAdd(&f[k].z, -dfk*dzk);
            }
		}
        if (type[i] == CHROM){

            float3 ri = r[i];
            int ks;
            for (ks = 0; ks < N_kt; ks ++){            
                if (i < N_pol*N_mt*N_mt_max + (ks + 1)*kn - chrom_bn/2 && i > N_pol*N_mt*N_mt_max + (ks + 1)*kn - chrom_bn || i < N_pol*N_mt*N_mt_max + (ks + 1)*kn && i > N_pol*N_mt*N_mt_max + (ks + 1)*kn - chrom_bn/2){
                    int k = i - 1;
			        float3 rk = r[k];

			        float dxk = rk.x - ri.x;
			        float dyk = rk.y - ri.y;
			        float dzk = rk.z - ri.z;

			        float drk = sqrtf(dxk*dxk + dyk*dyk + dzk*dzk);
			        float dfk = 200*K_kt*(drk - kt_radius[i - N_pol*N_mt*N_mt_max])*powf(drk, -1);
                    
                    //printf("%d\t%d\t%f\t%f\n", i, k, drk, kt_radius[i - N_pol*N_mt*N_mt_max]);
			        atomicAdd(&f[i].x, dfk*dxk);
			        atomicAdd(&f[i].y, dfk*dyk);		
			        atomicAdd(&f[i].z, dfk*dzk);

			        atomicAdd(&f[k].x, -dfk*dxk);
			        atomicAdd(&f[k].y, -dfk*dyk);		
			        atomicAdd(&f[k].z, -dfk*dzk);
                }
            }
        }
        if (type[i] == LEFT_NDC){
			float3 ri = r[i];
            int j, mult, ks;
            for (ks = 0; ks < N_kt; ks += 2){
                if (i < N_pol*N_mt*N_mt_max + (ks + 1)*kn && i >= N_pol*N_mt*N_mt_max + ks*kn){
                    j = N_pol*N_mt*N_mt_max + (ks + 1)*kn - 1 - chrom_bn;
                    mult = ks;
                }
            }
            float3 rj = r[j];

			float dx = rj.x - ri.x;
			float dy = rj.y - ri.y;
			float dz = rj.z - ri.z;

			float dr = sqrtf(dx*dx + dy*dy + dz*dz);
			float df = K_kt*(dr - kt_radius[i - N_pol*N_mt*N_mt_max])*powf(dr, -1);

			atomicAdd(&f[i].x, df*dx);
			atomicAdd(&f[i].y, df*dy);		
			atomicAdd(&f[i].z, df*dz);
            
            int l;          
            for (l = 0; l < harmonicKinCount[i - N_pol*N_mt*N_mt_max]; l++){
				float3 rl = r[N_pol*N_mt*N_mt_max + kn*mult + harmonicKin[(i - N_pol*N_mt*N_mt_max)*MpK + l]];

                float dxl = rl.x - ri.x;
				float dyl = rl.y - ri.y;
				float dzl = rl.z - ri.z;	
				float drl = sqrtf(dxl*dxl + dyl*dyl + dzl*dzl);
               
				float dfl = K_kt*(drl - harmonicKinRadii[(i - N_pol*N_mt*N_mt_max)*MpK + l])*powf(drl, -1)/100;
                             
                atomicAdd(&f[i].x, dfl*dxl);
			    atomicAdd(&f[i].y, dfl*dyl);		
			    atomicAdd(&f[i].z, dfl*dzl);

                atomicAdd(&f[N_pol*N_mt*N_mt_max + kn*mult + harmonicKin[(i - N_pol*N_mt*N_mt_max)*MpK + l]].x, -dfl*dxl);
			    atomicAdd(&f[N_pol*N_mt*N_mt_max + kn*mult + harmonicKin[(i - N_pol*N_mt*N_mt_max)*MpK + l]].y, -dfl*dyl);	
			    atomicAdd(&f[N_pol*N_mt*N_mt_max + kn*mult + harmonicKin[(i - N_pol*N_mt*N_mt_max)*MpK + l]].z, -dfl*dzl);
            }
        }
        if (type[i] == SHELL_LEFT){
			float3 ri = r[i];
            int j, mult, ks;
            for (ks = 0; ks < N_kt; ks += 2){
                if (i < N_pol*N_mt*N_mt_max + (ks + 1)*kn && i >= N_pol*N_mt*N_mt_max + ks*kn){
                    j = N_pol*N_mt*N_mt_max + (ks + 1)*kn - 1 - chrom_bn;
                    mult = ks;
                }
            }

            float3 rj = r[j];

			float dx = rj.x - ri.x;
			float dy = rj.y - ri.y;
			float dz = rj.z - ri.z;

			float dr = sqrtf(dx*dx + dy*dy + dz*dz);
			float df = K_kt*(dr - kt_radius[i - N_pol*N_mt*N_mt_max])*powf(dr, -1);

			atomicAdd(&f[i].x, df*dx);
			atomicAdd(&f[i].y, df*dy);		
			atomicAdd(&f[i].z, df*dz);
            
            int l;          
            for (l = 0; l < harmonicKinCount[i - N_pol*N_mt*N_mt_max]; l++){
				float3 rl = r[N_pol*N_mt*N_mt_max + kn*mult + harmonicKin[(i - N_pol*N_mt*N_mt_max)*MpK + l]];

                float dxl = rl.x - ri.x;
				float dyl = rl.y - ri.y;
				float dzl = rl.z - ri.z;	
				float drl = sqrtf(dxl*dxl + dyl*dyl + dzl*dzl);
               
				float dfl = K_kt*(drl - harmonicKinRadii[(i - N_pol*N_mt*N_mt_max)*MpK + l])*powf(drl, -1);
                             
                atomicAdd(&f[i].x, dfl*dxl);
			    atomicAdd(&f[i].y, dfl*dyl);		
			    atomicAdd(&f[i].z, dfl*dzl);

                atomicAdd(&f[N_pol*N_mt*N_mt_max + kn*mult + harmonicKin[(i - N_pol*N_mt*N_mt_max)*MpK + l]].x, -dfl*dxl);
			    atomicAdd(&f[N_pol*N_mt*N_mt_max + kn*mult + harmonicKin[(i - N_pol*N_mt*N_mt_max)*MpK + l]].y, -dfl*dyl);	
			    atomicAdd(&f[N_pol*N_mt*N_mt_max + kn*mult + harmonicKin[(i - N_pol*N_mt*N_mt_max)*MpK + l]].z, -dfl*dzl);
            }
        }
        if (type[i] == RIGHT_NDC){
			float3 ri = r[i];
            int j, mult, ks;
            for (ks = 1; ks < N_kt; ks += 2){
                if (i < N_pol*N_mt*N_mt_max + (ks + 1)*kn && i >= N_pol*N_mt*N_mt_max + ks*kn){
                    j = N_pol*N_mt*N_mt_max + (ks + 1)*kn - 1 - chrom_bn;
                    mult = ks;
                }
            }
            float3 rj = r[j];

			float dx = rj.x - ri.x;
			float dy = rj.y - ri.y;
			float dz = rj.z - ri.z;

			float dr = sqrtf(dx*dx + dy*dy + dz*dz);
			float df = K_kt*(dr - kt_radius[i - N_pol*N_mt*N_mt_max])*powf(dr, -1);
            //printf("%d\t%f\t%f\t%f\n", i, dr, kt_radius[i - N_pol*N_mt*N_mt_max], df);

			atomicAdd(&f[i].x, df*dx);
			atomicAdd(&f[i].y, df*dy);		
			atomicAdd(&f[i].z, df*dz);

            int l;
            for (l = 0; l < harmonicKinCount[i - N_pol*N_mt*N_mt_max]; l++){
			    float3 rl = r[N_pol*N_mt*N_mt_max + kn*mult + harmonicKin[(i - N_pol*N_mt*N_mt_max)*MpK + l]];
                float dxl = rl.x - ri.x;
			    float dyl = rl.y - ri.y;
			    float dzl = rl.z - ri.z;	
			    float drl = sqrtf(dxl*dxl + dyl*dyl + dzl*dzl);

			    float dfl = K_kt*(drl - harmonicKinRadii[(i - N_pol*N_mt*N_mt_max)*MpK + l])*powf(drl, -1)/100;
                //printf("%d\t%f\t%f\n", i, drl, harmonicKinRadii[(i - N_pol*N_mt*N_mt_max)*MpK + l]);                
                atomicAdd(&f[i].x, dfl*dxl);
		        atomicAdd(&f[i].y, dfl*dyl);		
		        atomicAdd(&f[i].z, dfl*dzl);

                atomicAdd(&f[N_pol*N_mt*N_mt_max + kn*mult + harmonicKin[(i - N_pol*N_mt*N_mt_max)*MpK + l]].x, -dfl*dxl);
		        atomicAdd(&f[N_pol*N_mt*N_mt_max + kn*mult + harmonicKin[(i - N_pol*N_mt*N_mt_max)*MpK + l]].y, -dfl*dyl);		
		        atomicAdd(&f[N_pol*N_mt*N_mt_max + kn*mult + harmonicKin[(i - N_pol*N_mt*N_mt_max)*MpK + l]].z, -dfl*dzl);
            }
        }
        if (type[i] == SHELL_RIGHT){
			float3 ri = r[i];
            int j, mult, ks;
            for (ks = 1; ks < N_kt; ks += 2){
                if (i < N_pol*N_mt*N_mt_max + (ks + 1)*kn && i >= N_pol*N_mt*N_mt_max + ks*kn){
                    j = N_pol*N_mt*N_mt_max + (ks + 1)*kn - 1 - chrom_bn;
                    mult = ks;
                }
            }
            float3 rj = r[j];

			float dx = rj.x - ri.x;
			float dy = rj.y - ri.y;
			float dz = rj.z - ri.z;

			float dr = sqrtf(dx*dx + dy*dy + dz*dz);
			float df = K_kt*(dr - kt_radius[i - N_pol*N_mt*N_mt_max])*powf(dr, -1);
            //printf("%d\t%f\t%f\t%f\n", i, dr, kt_radius[i - N_pol*N_mt*N_mt_max], df);

			atomicAdd(&f[i].x, df*dx);
			atomicAdd(&f[i].y, df*dy);		
			atomicAdd(&f[i].z, df*dz);

            int l;
            for (l = 0; l < harmonicKinCount[i - N_pol*N_mt*N_mt_max]; l++){
			    float3 rl = r[N_pol*N_mt*N_mt_max + kn*mult + harmonicKin[(i - N_pol*N_mt*N_mt_max)*MpK + l]];
                float dxl = rl.x - ri.x;
			    float dyl = rl.y - ri.y;
			    float dzl = rl.z - ri.z;	
			    float drl = sqrtf(dxl*dxl + dyl*dyl + dzl*dzl);

			    float dfl = K_kt*(drl - harmonicKinRadii[(i - N_pol*N_mt*N_mt_max)*MpK + l])*powf(drl, -1);
                //printf("%d\t%f\t%f\n", i, drl, harmonicKinRadii[(i - N_pol*N_mt*N_mt_max)*MpK + l]);                
                atomicAdd(&f[i].x, dfl*dxl);
		        atomicAdd(&f[i].y, dfl*dyl);		
		        atomicAdd(&f[i].z, dfl*dzl);

                atomicAdd(&f[N_pol*N_mt*N_mt_max + kn*mult + harmonicKin[(i - N_pol*N_mt*N_mt_max)*MpK + l]].x, -dfl*dxl);
		        atomicAdd(&f[N_pol*N_mt*N_mt_max + kn*mult + harmonicKin[(i - N_pol*N_mt*N_mt_max)*MpK + l]].y, -dfl*dyl);		
		        atomicAdd(&f[N_pol*N_mt*N_mt_max + kn*mult + harmonicKin[(i - N_pol*N_mt*N_mt_max)*MpK + l]].z, -dfl*dzl);
            }
        }
    //	__syncthreads();
	}
}
