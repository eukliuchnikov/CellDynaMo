#include "hip/hip_runtime.h"
#include "langevin.cuh"
#include "../Chemistry/react.h"
#include "../Math/ht.cu"

__global__ void integrateGPU(float3* r, float3* f, int N, int* type, int kn, float3 l_f, float3 r_f, Param* d_parameters){
	int i;
    float time_step = d_parameters->timestep;
    float viscos = d_parameters->visc;
    float temperatur = d_parameters->temper;
	i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < N){
        //float4 rf = rforce(i);
        //float pole_gamma = 6*M_PI*viscos*d_parameters->pole_r;
        //float pole_var = sqrtf(KB*temperatur*2.0*time_step*pow(pole_gamma, -1));
        float kt_gamma = 6*M_PI*viscos*d_parameters->ndc_d/2.0;
        //float kt_var = sqrtf(KB*temperatur*2.0*time_step*pow(kt_gamma, -1));
        float ktc_gamma = 6*M_PI*viscos*d_parameters->kt_r;
        float shell_gamma = 6*M_PI*viscos*60.0;
        float ktc_var = sqrtf(KB*temperatur*2.0*time_step*pow(ktc_gamma, -1));
//MT dynamics
		if (type[i] == MT_REG || type[i] == PLUS_DET || type[i] == PLUS_ATT || type[i] == PLUS_DET_INVALID){
            float gamma = 6*M_PI*viscos*d_parameters->mt_r;
            float var = sqrtf(KB*temperatur*2.0*time_step*pow(gamma, -1));
			r[i].x += f[i].x*time_step*pow(gamma, -1)/* + var*rf.x*/;
			r[i].y += f[i].y*time_step*pow(gamma, -1);
			r[i].z += f[i].z*time_step*pow(gamma, -1);        
		}
//Kinetochores
		if (type[i] == LEFT_NDC || type[i] == RIGHT_NDC){
			r[i].x += f[i].x*time_step*pow(kt_gamma, -1);
			r[i].y += f[i].y*time_step*pow(kt_gamma, -1);
			r[i].z += f[i].z*time_step*pow(kt_gamma, -1);            
		}

		else	if (type[i] == LEFT_KT || type[i] == RIGHT_KT || type[i] == CHROM){
			r[i].x += f[i].x*time_step*pow(ktc_gamma, -1);/* + ktc_var*rf.x*///;
			r[i].y += f[i].y*time_step*pow(ktc_gamma, -1);
			r[i].z += f[i].z*time_step*pow(ktc_gamma, -1);
		}
        else	if (type[i] == SHELL_LEFT || type[i] == SHELL_RIGHT){
			r[i].x += f[i].x*time_step*pow(shell_gamma, -1);/* + ktc_var*rf.x*///;
			r[i].y += f[i].y*time_step*pow(shell_gamma, -1);
			r[i].z += f[i].z*time_step*pow(shell_gamma, -1);
		}
		/*else	if (type[i] == LEFT_POLE || type[i] == LEFT_MINUS){
			r[i].x += l_f.x;
			r[i].y += l_f.y;
			r[i].z += l_f.z;
        }
        else	if (type[i] == RIGHT_POLE || type[i] == RIGHT_MINUS){
			r[i].x += r_f.x;
			r[i].y += r_f.y;
			r[i].z += r_f.z;
        }*/
        f[i].x = 0.0f;
    	f[i].y = 0.0f;
	    f[i].z = 0.0f;
	   // __syncthreads();
	}
}

__global__ void CoM(float3* r, float3* f, int N, int* type, int kn){
	int i;
	i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < N){
//MT dynamics
//Kinetochores
		if (type[i] == LEFT_KT){
			float dx = (r[i].x + r[i + kn].x)/2.0;
			float dy = (r[i].y + r[i + kn].y)/2.0;
			float dz = (r[i].z + r[i + kn].z)/2.0;
            r[i].x -= dx;
            r[i].y -= dy;
            r[i].z -= dz;
            r[i + kn].x -= dx;
            r[i + kn].y -= dy;
            r[i + kn].z -= dz;            
		}
	   // __syncthreads();
	}
}

__global__ void integrateGPU_init(float3* r, float3* f, int N, int* type, int kn, Param* d_parameters){
	int i;
    float time_step = d_parameters->timestep;
    float viscos = d_parameters->visc;
    float temperatur = d_parameters->temper;
	i = blockIdx.x*blockDim.x + threadIdx.x;
    //float mult = 
	if (i < N){
        float kt_gamma = 6*M_PI*viscos*d_parameters->ndc_d/2.0;
        //float kt_var = sqrtf(KB*temperatur*2.0*time_step*pow(kt_gamma, -1));
        float ktc_gamma = 6*M_PI*viscos*d_parameters->kt_r;
        //float ktc_var = sqrtf(KB*temperatur*2.0*time_step*pow(ktc_gamma, -1));
		if (type[i] == LEFT_NDC || type[i] == RIGHT_NDC){
            
			r[i].x += f[i].x*time_step*10*pow(kt_gamma, -1);
			r[i].y += f[i].y*time_step*10*pow(kt_gamma, -1);
			r[i].z += f[i].z*time_step*10*pow(kt_gamma, -1);            
		}

		else	if (type[i] == LEFT_KT || type[i] == RIGHT_KT || type[i] == CHROM){
            if (f[i].x > 1000000000.0){
                f[i].x = 1000000000.0;
            }
            if (f[i].y > 1000000000.0){
                f[i].y = 1000000000.0;
            }
            if (f[i].z > 1000000000.0){
                f[i].z = 1000000000.0;
            }
            if (f[i].x < -1000000000.0){
                f[i].x = -1000000000.0;
            }
            if (f[i].y < -1000000000.0){
                f[i].y = -1000000000.0;
            }
            if (f[i].z < -1000000000.0){
                f[i].z = -1000000000.0;
            }
			r[i].x += f[i].x*time_step*10*pow(ktc_gamma, -1);/* + ktc_var*rf.x*///;
			r[i].y += f[i].y*time_step*10*pow(ktc_gamma, -1);
            r[i].z += f[i].z*time_step*10*pow(ktc_gamma, -1);
		}
        f[i].x = 0.0f;
    	f[i].y = 0.0f;
	    f[i].z = 0.0f;
	   // __syncthreads();
	}
}

float3 kin_force(int index, float3* f, Param* d_parameters){
    float3 result;
    float time_step = d_parameters->timestep;
    float viscos = d_parameters->visc;
    float temperatur = d_parameters->temper;
    float ktc_gamma = 6*M_PI*viscos*d_parameters->kt_r;
    float ktc_var = sqrtf(KB*temperatur*2.0*time_step*pow(ktc_gamma, -1));
    result.x = f[index].x*time_step*pow(ktc_gamma, -1);
    result.y = f[index].y*time_step*pow(ktc_gamma, -1);
    result.z = f[index].z*time_step*pow(ktc_gamma, -1)/* - ktc_var*gasdev(&seed)*/;
    f[index].x = 0.0f;
    f[index].y = 0.0f;
    f[index].z = 0.0f;
    return result;
}

float3 pole_force(int index, float3* f, Param* d_parameters){
    float3 result;
    float time_step = d_parameters->timestep;
    float viscos = d_parameters->visc;
    float temperatur = d_parameters->temper;
    float pole_gamma = 6*M_PI*viscos*d_parameters->pole_r;
    float pole_var = sqrtf(KB*temperatur*2.0*time_step*pow(pole_gamma, -1));
    result.x = f[index].x*time_step*pow(pole_gamma, -1)/* - pole_var*gasdev(&seed)*/;
    result.y = f[index].y*time_step*pow(pole_gamma, -1);
    result.z = f[index].z*time_step*pow(pole_gamma, -1);
    f[index].x = 0.0f;
    f[index].y = 0.0f;
    f[index].z = 0.0f;
    return result;
}

void checkCUDAError(){
	hipError_t error = hipGetLastError();
	if(error != hipSuccess){
		printf("CUDA error: %s \n", hipGetErrorString(error));
exit(0);
	}
}

void KT_bonds_init(float* d_kin_cos, float* kin_cos, int* d_harmonicKinCount, int* a_harmonicKinCount, float* d_harmonicKinRadii, float* a_harmonicKinRadii, int* d_harmonicKin, int* a_harmonicKin, Param* d_parameters){
    int N_kt = d_parameters->kin_n;
    hipMalloc((void**)&d_kin_cos, N_kt*kn*sizeof(float));
    hipMemcpy(d_kin_cos, kin_cos, N_kt*kn*sizeof(float), hipMemcpyHostToDevice);

    hipMalloc((void**)&d_harmonicKinCount, N_kt*kn*sizeof(int));
    hipMemcpy(d_harmonicKinCount, a_harmonicKinCount, N_kt*kn*sizeof(int), hipMemcpyHostToDevice);
    hipMalloc((void**)&d_harmonicKinRadii, N_kt*kn*maxHarmonicKinPerMonomer*sizeof(float));
    hipMemcpy(d_harmonicKinRadii, a_harmonicKinRadii, N_kt*kn*maxHarmonicKinPerMonomer*sizeof(float), hipMemcpyHostToDevice);
    hipMalloc((void**)&d_harmonicKin, N_kt*kn*maxHarmonicKinPerMonomer*sizeof(int));
    hipMemcpy(d_harmonicKin, a_harmonicKin, N_kt*kn*maxHarmonicKinPerMonomer*sizeof(int), hipMemcpyHostToDevice);
}

void cudaRANDinit(){
   initRand(seed, mds.N);
}

void dynamics(DCD dcd, float time){
	long long int step;
    int blockSize = 512;
	int blockNum = mds.N/blockSize + 1;
    hipMemcpy(mds.d_type, mds.h_type, mds.N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(mds.d_connector, mds.h_connector, mds.N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(mds.d_att_l, mds.h_att_l, mds.N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(mds.d_link_l, mds.h_link_l, mds.N*sizeof(float), hipMemcpyHostToDevice);
    for (step = 0; step < 20; step++){
        //printf("%d\n", step);
	    //DCD_step(dcd);
        membrane(mds.h_f, mds.h_length);
		hipMemcpy(mds.d_r, mds.h_r, mds.N*sizeof(float3), hipMemcpyHostToDevice);	
        hipMemcpy(mds.d_f, mds.h_f, mds.N*sizeof(float3), hipMemcpyHostToDevice);
        hipMemcpy(mds.d_length, mds.h_length, mds.N*sizeof(float), hipMemcpyHostToDevice);
        //printf("Harmonic\n");       
        computeHarmonic<<<blockNum, blockSize>>>(mds.d_r, mds.d_f, mds.N, mds.d_type, kn, rk, d_harmonicKinCount, d_harmonicKin, d_harmonicKinRadii, d_kin_rad, d_parameters, mds.d_length);
        //printf("Bending\n");
        computeAngles<<<blockNum, blockSize>>>(mds.d_r, mds.d_f, mds.N, mds.d_type, kn, d_kin_cos, d_dyn_cos, d_parameters, mds.d_length);
        hipMemcpy(mds.d_att_n, mds.h_att_n, mds.N*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(mds.d_att_f, mds.h_att_f, mds.N*sizeof(float), hipMemcpyHostToDevice);   
        //printf("Pushing\n");      
        pushing<<<blockNum, blockSize>>>(mds.d_r, mds.d_f, mds.N, mds.d_type, kn, step, mds.d_att_n, mds.d_att_f, d_parameters);
        //printf("Pulling\n");
        pulling<<<blockNum, blockSize>>>(mds.d_r, mds.d_f, mds.N, mds.d_type, kn, step, mds.d_att_n, mds.d_att_f, d_parameters, mds.d_connector, mds.d_length, mds.d_att_l, mds.d_link_l);
        excl_vol_kt<<<blockNum, blockSize>>>(mds.d_r, mds.d_f, mds.N, mds.d_type, kn, d_parameters);
        hipMemcpy(mds.h_att_n, mds.d_att_n, mds.N*sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(mds.h_att_f, mds.d_att_f, mds.N*sizeof(float), hipMemcpyDeviceToHost);     
        for (int ks = 0; ks < h_parameters->kin_n; ks ++){
            if (step == 0 && (mds.h_att_n[ks*5] != 0)){
                printf("KT#%d FORCES:\tN_total\tN_pull\tN_push\tF_pull\tF_push\n%f\t%d\t%d\t%d\t%f\t%f\n", ks, time, mds.h_att_n[ks*5], mds.h_att_n[ks*5 + 1], mds.h_att_n[ks*5 + 2], mds.h_att_f[ks*5], mds.h_att_f[ks*5 + 1]);
            }
            mds.h_att_n[ks*5] = 0;
            mds.h_att_n[ks*5 + 1] = 0;
            mds.h_att_f[ks*5] = 0.0;
            mds.h_att_f[ks*5 + 1] = 0.0;
            mds.h_att_n[ks*5 + 2] = 0;
            /*mds.h_att_n[ks*5 + 3] = 0;
            mds.h_att_n[ks*5 + 4] = 0;
            mds.h_att_n[ks*5 + 5] = 0;*/
        } 
        int ln = mds.N - 2;
        int rn = mds.N - 1;  

        float3 left_pole = pole_force(ln, mds.h_f, h_parameters);
        float3 right_pole = pole_force(rn, mds.h_f, h_parameters);
        //printf("Integr\n");
		integrateGPU<<<blockNum, blockSize>>>(mds.d_r, mds.d_f, mds.N, mds.d_type, kn, left_pole, right_pole, d_parameters);
        //CoM<<<blockNum, blockSize>>>(mds.d_r, mds.d_f, mds.N, mds.d_type, kn);
        hipMemcpy(mds.h_f, mds.d_f, mds.N*sizeof(float3), hipMemcpyDeviceToHost);
        hipMemcpy(mds.h_r, mds.d_r, mds.N*sizeof(float3), hipMemcpyDeviceToHost);
        hipMemcpy(mds.h_length, mds.d_length, mds.N*sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(dyn_cos, d_dyn_cos, KIN_COUNT*kn*sizeof(int), hipMemcpyDeviceToHost);
        checkCUDAError();
	}
}

void minimiz(DCD dcd){
	long long int step;
    int blockSize = 512;
	int blockNum = mds.N/blockSize + 1;
    hipMemcpy(mds.d_type, mds.h_type, mds.N*sizeof(int), hipMemcpyHostToDevice);
    for (step = 0; step < 30000; step++){
        if (step % 10000 == 0){
	        DCD_step(dcd);
        }
        membrane(mds.h_f, mds.h_length);
		hipMemcpy(mds.d_r, mds.h_r, mds.N*sizeof(float3), hipMemcpyHostToDevice);	
        hipMemcpy(mds.d_f, mds.h_f, mds.N*sizeof(float3), hipMemcpyHostToDevice);
        computeHarmonic<<<blockNum, blockSize>>>(mds.d_r, mds.d_f, mds.N, mds.d_type, kn, rk, d_harmonicKinCount, d_harmonicKin, d_harmonicKinRadii, d_kin_rad, d_parameters, mds.d_length);
        computeAngles<<<blockNum, blockSize>>>(mds.d_r, mds.d_f, mds.N, mds.d_type, kn, d_kin_cos, d_dyn_cos, d_parameters, mds.d_length);
        hipMemcpy(mds.d_att_n, mds.h_att_n, mds.N*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(mds.d_att_f, mds.h_att_f, mds.N*sizeof(float), hipMemcpyHostToDevice);   
        pushing<<<blockNum, blockSize>>>(mds.d_r, mds.d_f, mds.N, mds.d_type, kn, step, mds.d_att_n, mds.d_att_f, d_parameters);
        excl_vol_kt<<<blockNum, blockSize>>>(mds.d_r, mds.d_f, mds.N, mds.d_type, kn, d_parameters);
        float radi = CH_TER;
        chrom_ter<<<blockNum, blockSize>>>(mds.d_r, mds.d_f, mds.N, mds.d_type, kn, d_parameters, radi);
		integrateGPU_init<<<blockNum, blockSize>>>(mds.d_r, mds.d_f, mds.N, mds.d_type, kn, d_parameters);
        hipMemcpy(mds.h_f, mds.d_f, mds.N*sizeof(float3), hipMemcpyDeviceToHost);
        hipMemcpy(mds.h_r, mds.d_r, mds.N*sizeof(float3), hipMemcpyDeviceToHost);
        hipMemcpy(mds.h_length, mds.d_length, mds.N*sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(dyn_cos, d_dyn_cos, KIN_COUNT*kn*sizeof(int), hipMemcpyDeviceToHost);
        checkCUDAError();
	}
    for (step = 0; step < 30000; step++){
        if (step % 10000 == 0){
	        DCD_step(dcd);
        }
        membrane(mds.h_f, mds.h_length);
		hipMemcpy(mds.d_r, mds.h_r, mds.N*sizeof(float3), hipMemcpyHostToDevice);	
        hipMemcpy(mds.d_f, mds.h_f, mds.N*sizeof(float3), hipMemcpyHostToDevice);
        computeHarmonic<<<blockNum, blockSize>>>(mds.d_r, mds.d_f, mds.N, mds.d_type, kn, rk, d_harmonicKinCount, d_harmonicKin, d_harmonicKinRadii, d_kin_rad, d_parameters, mds.d_length);
        computeAngles<<<blockNum, blockSize>>>(mds.d_r, mds.d_f, mds.N, mds.d_type, kn, d_kin_cos, d_dyn_cos, d_parameters, mds.d_length);
        hipMemcpy(mds.d_att_n, mds.h_att_n, mds.N*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(mds.d_att_f, mds.h_att_f, mds.N*sizeof(float), hipMemcpyHostToDevice);   
        pushing<<<blockNum, blockSize>>>(mds.d_r, mds.d_f, mds.N, mds.d_type, kn, step, mds.d_att_n, mds.d_att_f, d_parameters);
        excl_vol_kt<<<blockNum, blockSize>>>(mds.d_r, mds.d_f, mds.N, mds.d_type, kn, d_parameters);
        float radi = 0.625*CH_TER;
        chrom_ter<<<blockNum, blockSize>>>(mds.d_r, mds.d_f, mds.N, mds.d_type, kn, d_parameters, radi);
		integrateGPU_init<<<blockNum, blockSize>>>(mds.d_r, mds.d_f, mds.N, mds.d_type, kn, d_parameters);
        hipMemcpy(mds.h_f, mds.d_f, mds.N*sizeof(float3), hipMemcpyDeviceToHost);
        hipMemcpy(mds.h_r, mds.d_r, mds.N*sizeof(float3), hipMemcpyDeviceToHost);
        hipMemcpy(mds.h_length, mds.d_length, mds.N*sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(dyn_cos, d_dyn_cos, KIN_COUNT*kn*sizeof(int), hipMemcpyDeviceToHost);
        checkCUDAError();
	}
    for (step = 0; step < 30000; step++){
        if (step % 10000 == 0){
	        DCD_step(dcd);
        }
        membrane(mds.h_f, mds.h_length);
		hipMemcpy(mds.d_r, mds.h_r, mds.N*sizeof(float3), hipMemcpyHostToDevice);	
        hipMemcpy(mds.d_f, mds.h_f, mds.N*sizeof(float3), hipMemcpyHostToDevice);
        computeHarmonic<<<blockNum, blockSize>>>(mds.d_r, mds.d_f, mds.N, mds.d_type, kn, rk, d_harmonicKinCount, d_harmonicKin, d_harmonicKinRadii, d_kin_rad, d_parameters, mds.d_length);
        computeAngles<<<blockNum, blockSize>>>(mds.d_r, mds.d_f, mds.N, mds.d_type, kn, d_kin_cos, d_dyn_cos, d_parameters, mds.d_length);
        hipMemcpy(mds.d_att_n, mds.h_att_n, mds.N*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(mds.d_att_f, mds.h_att_f, mds.N*sizeof(float), hipMemcpyHostToDevice);   
        pushing<<<blockNum, blockSize>>>(mds.d_r, mds.d_f, mds.N, mds.d_type, kn, step, mds.d_att_n, mds.d_att_f, d_parameters);
        excl_vol_kt<<<blockNum, blockSize>>>(mds.d_r, mds.d_f, mds.N, mds.d_type, kn, d_parameters);
        float radi = CH_TER;
        chrom_ter<<<blockNum, blockSize>>>(mds.d_r, mds.d_f, mds.N, mds.d_type, kn, d_parameters, radi);
		integrateGPU_init<<<blockNum, blockSize>>>(mds.d_r, mds.d_f, mds.N, mds.d_type, kn, d_parameters);
        hipMemcpy(mds.h_f, mds.d_f, mds.N*sizeof(float3), hipMemcpyDeviceToHost);
        hipMemcpy(mds.h_r, mds.d_r, mds.N*sizeof(float3), hipMemcpyDeviceToHost);
        hipMemcpy(mds.h_length, mds.d_length, mds.N*sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(dyn_cos, d_dyn_cos, KIN_COUNT*kn*sizeof(int), hipMemcpyDeviceToHost);
        checkCUDAError();
	}
}
