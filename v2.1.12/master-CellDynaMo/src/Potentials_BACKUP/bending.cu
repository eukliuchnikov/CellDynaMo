#include "hip/hip_runtime.h"
#include "bending.cuh"		

__global__ void computeAngles(float3* r, float3* f, int N, int* type, int kn, float* kt_cos, float* dyn_cos, Param* d_parameters, float* length){
	int i;
    float temper = d_parameters->temper;
    float pers_l = d_parameters->lp;
    int N_pol = d_parameters->pole_n;
    int N_mt = d_parameters->mt_n;
    int N_kt = d_parameters->kin_n;
    int N_mt_max = d_parameters->max_mt_n;
    float K_kt = d_parameters->k_kt;
    int chrom_cond = d_parameters->chrom;
    int chrom_bn = chrom_cond*d_parameters->chrom_num;
	i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < N){
		if (type[i] == LEFT_MINUS){
            float k_theta = 10.0*KB*temper*pers_l/1000.0;
            float theta0 = M_PI;
            int j = i + 1;
            int k = N - 2;
			float3 r1 = r[j];
			float3 r2 = r[i];
			float3 r3 = r[k];

			float3 dr12, dr32;
			dr12.x = r1.x - r2.x;
			dr12.y = r1.y - r2.y;
			dr12.z = r1.z - r2.z;

			dr32.x = r3.x - r2.x;
			dr32.y = r3.y - r2.y;
			dr32.z = r3.z - r2.z;

			float r12inv = 1.0f*powf(dr12.x*dr12.x + dr12.y*dr12.y + dr12.z*dr12.z, -0.5);
			float r32inv = 1.0f*powf(dr32.x*dr32.x + dr32.y*dr32.y + dr32.z*dr32.z, -0.5);
			float costheta = (dr12.x*dr32.x + dr12.y*dr32.y + dr12.z*dr32.z)*r12inv*r32inv;
			if (costheta > 1.0f){
				costheta = 1.0f;
			}
			else	if (costheta < -1.0f){
				costheta = -1.0f;
			}
			float sintheta = sqrtf(1.0f - costheta*costheta);
			float theta = acos(costheta);
			float diff = theta - theta0;
   			if (sintheta < 1.e-6){
				if (diff < 0){
					diff *= 2.0f*k_theta;
				}
				else{
					diff *= -2.0f*k_theta;
				}
			}
			else{
				diff *= (-2.0f*k_theta)*powf(sintheta, -1);
			}
			float c1 = diff*r12inv;
			float c2 = diff*r32inv;

			float3 f1, f2, f3;
			f1.x = c1*(dr12.x*(r12inv*costheta) - dr32.x*r32inv);
			f1.y = c1*(dr12.y*(r12inv*costheta) - dr32.y*r32inv);
			f1.z = c1*(dr12.z*(r12inv*costheta) - dr32.z*r32inv);
			f2 = f1;
			f3.x = c2*(dr32.x*(r32inv*costheta) - dr12.x*r12inv);
			f3.y = c2*(dr32.y*(r32inv*costheta) - dr12.y*r12inv);
			f3.z = c2*(dr32.z*(r32inv*costheta) - dr12.z*r12inv);
			f2.x += f3.x;
			f2.y += f3.y;
			f2.z += f3.z;

			f2.x = -f2.x;
			f2.y = -f2.y;
			f2.z = -f2.z;

            atomicAdd(&f[j].x, f1.x);
		    atomicAdd(&f[j].y, f1.y);		
		    atomicAdd(&f[j].z, f1.z);

            /*atomicAdd(&f[i].x, f2.x);
			atomicAdd(&f[i].y, f2.y);		
			atomicAdd(&f[i].z, f2.z);*/
           
            atomicAdd(&f[k].x, f3.x);
		    atomicAdd(&f[k].y, f3.y);		
		    atomicAdd(&f[k].z, f3.z);       
		}
        if (type[i] == RIGHT_MINUS){
            float k_theta = 10.0*KB*temper*pers_l/1000.0;
            float theta0 = M_PI;
            int j = i + 1;
            int k = N - 1;
			float3 r1 = r[j];
			float3 r2 = r[i];
			float3 r3 = r[k];

			float3 dr12, dr32;
			dr12.x = r1.x - r2.x;
			dr12.y = r1.y - r2.y;
			dr12.z = r1.z - r2.z;

			dr32.x = r3.x - r2.x;
			dr32.y = r3.y - r2.y;
			dr32.z = r3.z - r2.z;

			float r12inv = 1.0f*powf(dr12.x*dr12.x + dr12.y*dr12.y + dr12.z*dr12.z, -0.5);
			float r32inv = 1.0f*powf(dr32.x*dr32.x + dr32.y*dr32.y + dr32.z*dr32.z, -0.5);
			float costheta = (dr12.x*dr32.x + dr12.y*dr32.y + dr12.z*dr32.z)*r12inv*r32inv;
			if (costheta > 1.0f){
				costheta = 1.0f;
			}
			else	if (costheta < -1.0f){
				costheta = -1.0f;
			}
			float sintheta = sqrtf(1.0f - costheta*costheta);
			float theta = acos(costheta);
			float diff = theta - theta0;
   			if (sintheta < 1.e-6){
				if (diff < 0){
					diff *= 2.0f*k_theta;
				}
				else{
					diff *= -2.0f*k_theta;
				}
			}
			else{
				diff *= (-2.0f*k_theta)*powf(sintheta, -1);
			}
			float c1 = diff*r12inv;
			float c2 = diff*r32inv;

			float3 f1, f2, f3;
			f1.x = c1*(dr12.x*(r12inv*costheta) - dr32.x*r32inv);
			f1.y = c1*(dr12.y*(r12inv*costheta) - dr32.y*r32inv);
			f1.z = c1*(dr12.z*(r12inv*costheta) - dr32.z*r32inv);
			f2 = f1;
			f3.x = c2*(dr32.x*(r32inv*costheta) - dr12.x*r12inv);
			f3.y = c2*(dr32.y*(r32inv*costheta) - dr12.y*r12inv);
			f3.z = c2*(dr32.z*(r32inv*costheta) - dr12.z*r12inv);
			f2.x += f3.x;
			f2.y += f3.y;
			f2.z += f3.z;

			f2.x = -f2.x;
			f2.y = -f2.y;
			f2.z = -f2.z;

            atomicAdd(&f[j].x, f1.x);
		    atomicAdd(&f[j].y, f1.y);		
		    atomicAdd(&f[j].z, f1.z);

            /*atomicAdd(&f[i].x, f2.x);
			atomicAdd(&f[i].y, f2.y);		
			atomicAdd(&f[i].z, f2.z);*/
           
            atomicAdd(&f[k].x, f3.x);
		    atomicAdd(&f[k].y, f3.y);		
		    atomicAdd(&f[k].z, f3.z);       
		}
        if (type[i] == MT_REG || type[i] == MT_END){
            float k_theta = 10.0*KB*temper*pers_l/1000.0;
            float theta0 = M_PI;
            int j = i + 1;
            int k = i - 1;
			float3 r1 = r[j];
			float3 r2 = r[i];
			float3 r3 = r[k];

			float3 dr12, dr32;
			dr12.x = r1.x - r2.x;
			dr12.y = r1.y - r2.y;
			dr12.z = r1.z - r2.z;

			dr32.x = r3.x - r2.x;
			dr32.y = r3.y - r2.y;
			dr32.z = r3.z - r2.z;

			float r12inv = 1.0f*powf(dr12.x*dr12.x + dr12.y*dr12.y + dr12.z*dr12.z, -0.5);
			float r32inv = 1.0f*powf(dr32.x*dr32.x + dr32.y*dr32.y + dr32.z*dr32.z, -0.5);
			float costheta = (dr12.x*dr32.x + dr12.y*dr32.y + dr12.z*dr32.z)*r12inv*r32inv;
			if (costheta > 1.0f){
				costheta = 1.0f;
			}
			else	if (costheta < -1.0f){
				costheta = -1.0f;
			}
			float sintheta = sqrtf(1.0f - costheta*costheta);
			float theta = acos(costheta);
			float diff = theta - theta0;
   			if (sintheta < 1.e-6){
				if (diff < 0){
					diff *= 2.0f*k_theta;
				}
				else{
					diff *= -2.0f*k_theta;
				}
			}
			else{
				diff *= (-2.0f*k_theta)*powf(sintheta, -1);
			}
			float c1 = diff*r12inv;
			float c2 = diff*r32inv;

			float3 f1, f2, f3;
			f1.x = c1*(dr12.x*(r12inv*costheta) - dr32.x*r32inv);
			f1.y = c1*(dr12.y*(r12inv*costheta) - dr32.y*r32inv);
			f1.z = c1*(dr12.z*(r12inv*costheta) - dr32.z*r32inv);
			f2 = f1;
			f3.x = c2*(dr32.x*(r32inv*costheta) - dr12.x*r12inv);
			f3.y = c2*(dr32.y*(r32inv*costheta) - dr12.y*r12inv);
			f3.z = c2*(dr32.z*(r32inv*costheta) - dr12.z*r12inv);
			f2.x += f3.x;
			f2.y += f3.y;
			f2.z += f3.z;

			f2.x = -f2.x;
			f2.y = -f2.y;
			f2.z = -f2.z;

            atomicAdd(&f[j].x, f1.x);
		    atomicAdd(&f[j].y, f1.y);		
		    atomicAdd(&f[j].z, f1.z);

            atomicAdd(&f[i].x, f2.x);
			atomicAdd(&f[i].y, f2.y);		
			atomicAdd(&f[i].z, f2.z);
            
            if (type[k] != RIGHT_MINUS && type[k] != LEFT_MINUS){
                atomicAdd(&f[k].x, f3.x);
		        atomicAdd(&f[k].y, f3.y);		
		        atomicAdd(&f[k].z, f3.z);       
            }
		}


        if ((type[i] == LEFT_KT || type[i] == RIGHT_KT) && chrom_bn != 0){
            float k_theta = K_kt;
            for (int chrom = 1; chrom < chrom_bn; chrom += chrom_bn/2){
                int j = i + chrom;
                int k = j + 1;

			    float3 r1 = r[k];
			    float3 r2 = r[j];
			    float3 r3 = r[i];

			    float3 dr12, dr32;
			    dr12.x = r1.x - r2.x;
			    dr12.y = r1.y - r2.y;
			    dr12.z = r1.z - r2.z;

			    dr32.x = r3.x - r2.x;
			    dr32.y = r3.y - r2.y;
			    dr32.z = r3.z - r2.z;

			    float r12inv = 1.0f*powf(dr12.x*dr12.x + dr12.y*dr12.y + dr12.z*dr12.z, -0.5);
			    float r32inv = 1.0f*powf(dr32.x*dr32.x + dr32.y*dr32.y + dr32.z*dr32.z, -0.5);
			    float costheta = (dr12.x*dr32.x + dr12.y*dr32.y + dr12.z*dr32.z)*r12inv*r32inv;
			    if (costheta > 1.0f){
				    costheta = 1.0f;
			    }
			    else	if (costheta < -1.0f){
				    costheta = -1.0f;
			    }
			    float sintheta = sqrtf(1.0f - costheta*costheta);
			    float theta = acos(costheta);
			    float diff = theta - kt_cos[j - N_pol*N_mt*N_mt_max];

                dyn_cos[k - N_pol*N_mt*N_mt_max] = theta*180/M_PI;
                //printf ("BETA: %d\t%f\t%f\n", k, theta*180/M_PI, kt_cos[k - N_pol*N_mt*N_mt_max]*180/M_PI);
       			if (sintheta < 1.e-6){
				    if (diff < 0){
					    diff *= 2.0f*357140;
				    }
				    else{
					    diff *= -2.0f*357140;
				    }
			    }
			    else{
				    diff *= (-2.0f*357140)*powf(sintheta, -1);
			    }
			    float c1 = diff*r12inv;
			    float c2 = diff*r32inv;

			    float3 f1, f2, f3;
			    f1.x = c1*(dr12.x*(r12inv*costheta) - dr32.x*r32inv);
			    f1.y = c1*(dr12.y*(r12inv*costheta) - dr32.y*r32inv);
			    f1.z = c1*(dr12.z*(r12inv*costheta) - dr32.z*r32inv);
			    f2 = f1;
			    f3.x = c2*(dr32.x*(r32inv*costheta) - dr12.x*r12inv);
			    f3.y = c2*(dr32.y*(r32inv*costheta) - dr12.y*r12inv);
			    f3.z = c2*(dr32.z*(r32inv*costheta) - dr12.z*r12inv);
			    f2.x += f3.x;
			    f2.y += f3.y;
			    f2.z += f3.z;

			    f2.x = -f2.x;
			    f2.y = -f2.y;
			    f2.z = -f2.z;

                atomicAdd(&f[k].x, f1.x);
		        atomicAdd(&f[k].y, f1.y);		
		        atomicAdd(&f[k].z, f1.z);

                atomicAdd(&f[j].x, f2.x);
			    atomicAdd(&f[j].y, f2.y);		
			    atomicAdd(&f[j].z, f2.z);
               
                atomicAdd(&f[i].x, f3.x);
		        atomicAdd(&f[i].y, f3.y);		
		        atomicAdd(&f[i].z, f3.z);
            }

            int j = i + 1;
            int k = i + chrom_bn/2 + 1;

		    float3 r1 = r[k];
		    float3 r2 = r[i];
		    float3 r3 = r[j];

		    float3 dr12, dr32;
		    dr12.x = r1.x - r2.x;
		    dr12.y = r1.y - r2.y;
		    dr12.z = r1.z - r2.z;

		    dr32.x = r3.x - r2.x;
		    dr32.y = r3.y - r2.y;
		    dr32.z = r3.z - r2.z;

		    float r12inv = 1.0f*powf(dr12.x*dr12.x + dr12.y*dr12.y + dr12.z*dr12.z, -0.5);
		    float r32inv = 1.0f*powf(dr32.x*dr32.x + dr32.y*dr32.y + dr32.z*dr32.z, -0.5);
		    float costheta = (dr12.x*dr32.x + dr12.y*dr32.y + dr12.z*dr32.z)*r12inv*r32inv;
		    if (costheta > 1.0f){
			    costheta = 1.0f;
		    }
		    else	if (costheta < -1.0f){
			    costheta = -1.0f;
		    }
		    float sintheta = sqrtf(1.0f - costheta*costheta);
            float new_sin = (dr12.x*dr32.y - dr12.y*dr32.x)*r12inv*r32inv;
		    float theta = acos(costheta);
		    float diff = theta - kt_cos[j - N_pol*N_mt*N_mt_max];
            if (new_sin >= 0.0){
                dyn_cos[j - N_pol*N_mt*N_mt_max] = theta*180/M_PI;
            }
            if (new_sin < 0.0){
                float val = 360.0 - theta*180/M_PI;
                dyn_cos[j - N_pol*N_mt*N_mt_max] = val;
            }                        
            //printf ("ALPHA: %d\t%f\t%f\n", j, theta*180/M_PI, kt_cos[j - N_pol*N_mt*N_mt_max]*180/M_PI);
   			if (sintheta < 1.e-6){
			    if (diff < 0){
				    diff *= 2.0f*357140;
			    }
			    else{
				    diff *= -2.0f*357140;
			    }
		    }
		    else{
			    diff *= (-2.0f*357140)*powf(sintheta, -1);
		    }
		    float c1 = diff*r12inv;
		    float c2 = diff*r32inv;

		    float3 f1, f2, f3;
		    f1.x = c1*(dr12.x*(r12inv*costheta) - dr32.x*r32inv);
		    f1.y = c1*(dr12.y*(r12inv*costheta) - dr32.y*r32inv);
		    f1.z = c1*(dr12.z*(r12inv*costheta) - dr32.z*r32inv);
		    f2 = f1;
		    f3.x = c2*(dr32.x*(r32inv*costheta) - dr12.x*r12inv);
		    f3.y = c2*(dr32.y*(r32inv*costheta) - dr12.y*r12inv);
		    f3.z = c2*(dr32.z*(r32inv*costheta) - dr12.z*r12inv);
		    f2.x += f3.x;
		    f2.y += f3.y;
		    f2.z += f3.z;

		    f2.x = -f2.x;
		    f2.y = -f2.y;
		    f2.z = -f2.z;

            atomicAdd(&f[k].x, f1.x);
	        atomicAdd(&f[k].y, f1.y);		
	        atomicAdd(&f[k].z, f1.z);

            atomicAdd(&f[i].x, f2.x);
		    atomicAdd(&f[i].y, f2.y);		
		    atomicAdd(&f[i].z, f2.z);
           
            atomicAdd(&f[j].x, f3.x);
	        atomicAdd(&f[j].y, f3.y);		
	        atomicAdd(&f[j].z, f3.z);
            //printf("%d\t%d\t%d\t%f\t%f\n", i, j, k, theta, kt_cos[j - N_pol*N_mt*N_mt_max]); */
        }
        if (type[i] == CHROM){
            int j, k, ks;
            for (ks = 0; ks < N_kt; ks++){
                if (i < N_pol*N_mt*N_mt_max + (ks + 1)*kn - chrom_bn/2 && i > N_pol*N_mt*N_mt_max + (ks + 1)*kn - chrom_bn + 1 || i < N_pol*N_mt*N_mt_max + (ks + 1)*kn && i > N_pol*N_mt*N_mt_max + (ks + 1)*kn - chrom_bn/2 + 1){
                    j = i - 1;
                    k = j - 1;
                    float3 r1 = r[k];
		            float3 r2 = r[j];
		            float3 r3 = r[i];
                  
		            float3 dr12, dr32;
		            dr12.x = r1.x - r2.x;
		            dr12.y = r1.y - r2.y;
		            dr12.z = r1.z - r2.z;

		            dr32.x = r3.x - r2.x;
		            dr32.y = r3.y - r2.y;
		            dr32.z = r3.z - r2.z;

		            float r12inv = 1.0f*powf(dr12.x*dr12.x + dr12.y*dr12.y + dr12.z*dr12.z, -0.5);
		            float r32inv = 1.0f*powf(dr32.x*dr32.x + dr32.y*dr32.y + dr32.z*dr32.z, -0.5);
		            float costheta = (dr12.x*dr32.x + dr12.y*dr32.y + dr12.z*dr32.z)*r12inv*r32inv;
		            if (costheta > 1.0f){
			            costheta = 1.0f;
		            }
		            else	if (costheta < -1.0f){
			            costheta = -1.0f;
		            }
		            float sintheta = sqrtf(1.0f - costheta*costheta);
		            float theta = acos(costheta);
		            float diff = theta - kt_cos[i - N_pol*N_mt*N_mt_max];
                    dyn_cos[i - N_pol*N_mt*N_mt_max] = theta*180/M_PI;
                    //printf ("BETA: %d\t%f\t%f\n", k, theta*180/M_PI, kt_cos[k - N_pol*N_mt*N_mt_max]*180/M_PI);
           			if (sintheta < 1.e-6){
			            if (diff < 0){
				            diff *= 2.0f*357140;
			            }
			            else{
				            diff *= -2.0f*357140;
			            }
		            }
		            else{
			            diff *= (-2.0f*357140)*powf(sintheta, -1);
		            }
		            float c1 = diff*r12inv;
		            float c2 = diff*r32inv;

		            float3 f1, f2, f3;
		            f1.x = c1*(dr12.x*(r12inv*costheta) - dr32.x*r32inv);
		            f1.y = c1*(dr12.y*(r12inv*costheta) - dr32.y*r32inv);
		            f1.z = c1*(dr12.z*(r12inv*costheta) - dr32.z*r32inv);
		            f2 = f1;
		            f3.x = c2*(dr32.x*(r32inv*costheta) - dr12.x*r12inv);
		            f3.y = c2*(dr32.y*(r32inv*costheta) - dr12.y*r12inv);
		            f3.z = c2*(dr32.z*(r32inv*costheta) - dr12.z*r12inv);
		            f2.x += f3.x;
		            f2.y += f3.y;
		            f2.z += f3.z;

		            f2.x = -f2.x;
		            f2.y = -f2.y;
		            f2.z = -f2.z;

                    atomicAdd(&f[k].x, f1.x);
	                atomicAdd(&f[k].y, f1.y);		
	                atomicAdd(&f[k].z, f1.z);

                    atomicAdd(&f[j].x, f2.x);
		            atomicAdd(&f[j].y, f2.y);		
		            atomicAdd(&f[j].z, f2.z);
                   
                    atomicAdd(&f[i].x, f3.x);
	                atomicAdd(&f[i].y, f3.y);		
	                atomicAdd(&f[i].z, f3.z);
                }
            }
        }

        if (type[i] == LEFT_NDC){
            int j, k, ks;
            for (ks = 0; ks < N_kt; ks += 2){
                if (i < N_pol*N_mt*N_mt_max + (ks + 1)*kn && i >= N_pol*N_mt*N_mt_max + ks*kn){
                    j = N_pol*N_mt*N_mt_max + (ks + 1)*kn - 1 - chrom_bn;
                    k = N_pol*N_mt*N_mt_max + (ks + 2)*kn - 1 - chrom_bn;
                }
            }
			float3 r1 = r[i];
			float3 r2 = r[j];
			float3 r3 = r[k];

			float3 dr12, dr32;
			dr12.x = r1.x - r2.x;
			dr12.y = r1.y - r2.y;
			dr12.z = r1.z - r2.z;

			dr32.x = r3.x - r2.x;
			dr32.y = r3.y - r2.y;
			dr32.z = r3.z - r2.z;

			float r12inv = 1.0f*powf(dr12.x*dr12.x + dr12.y*dr12.y + dr12.z*dr12.z, -0.5);
			float r32inv = 1.0f*powf(dr32.x*dr32.x + dr32.y*dr32.y + dr32.z*dr32.z, -0.5);
			float costheta = (dr12.x*dr32.x + dr12.y*dr32.y + dr12.z*dr32.z)*r12inv*r32inv;
			if (costheta > 1.0f){
				costheta = 1.0f;
			}
			else	if (costheta < -1.0f){
				costheta = -1.0f;
			}
			float sintheta = sqrtf(1.0f - costheta*costheta);
			float theta = acos(costheta);
			float diff = theta - kt_cos[i - N_pol*N_mt*N_mt_max]; 
            if (sintheta < 1.e-6){
				if (diff < 0){
					diff *= 2.0f*K_kt*100000;
				}
				else{
					diff *= -2.0f*K_kt*100000;
				}
			}
			else{
				diff *= (-2.0f*K_kt*100000)*powf(sintheta, -1);
			}
			float c1 = diff*r12inv;
			float c2 = diff*r32inv;
    
			float3 f1, f2, f3;
			f1.x = c1*(dr12.x*(r12inv*costheta) - dr32.x*r32inv);
			f1.y = c1*(dr12.y*(r12inv*costheta) - dr32.y*r32inv);
			f1.z = c1*(dr12.z*(r12inv*costheta) - dr32.z*r32inv);
			f2 = f1;
			f3.x = c2*(dr32.x*(r32inv*costheta) - dr12.x*r12inv);
			f3.y = c2*(dr32.y*(r32inv*costheta) - dr12.y*r12inv);
			f3.z = c2*(dr32.z*(r32inv*costheta) - dr12.z*r12inv);
			f2.x += f3.x;
			f2.y += f3.y;
			f2.z += f3.z;

			f2.x = -f2.x;
			f2.y = -f2.y;
			f2.z = -f2.z;

            atomicAdd(&f[i].x, f1.x);
			atomicAdd(&f[i].y, f1.y);		
			atomicAdd(&f[i].z, f1.z);
        }
        if (type[i] == SHELL_LEFT){
            int j, k, ks;
            for (ks = 0; ks < N_kt; ks += 2){
                if (i < N_pol*N_mt*N_mt_max + (ks + 1)*kn && i >= N_pol*N_mt*N_mt_max + ks*kn){
                    j = N_pol*N_mt*N_mt_max + (ks + 1)*kn - 1 - chrom_bn;
                    k = N_pol*N_mt*N_mt_max + (ks + 2)*kn - 1 - chrom_bn;
                }
            }
			float3 r1 = r[i];
			float3 r2 = r[j];
			float3 r3 = r[k];

			float3 dr12, dr32;
			dr12.x = r1.x - r2.x;
			dr12.y = r1.y - r2.y;
			dr12.z = r1.z - r2.z;

			dr32.x = r3.x - r2.x;
			dr32.y = r3.y - r2.y;
			dr32.z = r3.z - r2.z;

			float r12inv = 1.0f*powf(dr12.x*dr12.x + dr12.y*dr12.y + dr12.z*dr12.z, -0.5);
			float r32inv = 1.0f*powf(dr32.x*dr32.x + dr32.y*dr32.y + dr32.z*dr32.z, -0.5);
			float costheta = (dr12.x*dr32.x + dr12.y*dr32.y + dr12.z*dr32.z)*r12inv*r32inv;
			if (costheta > 1.0f){
				costheta = 1.0f;
			}
			else	if (costheta < -1.0f){
				costheta = -1.0f;
			}
			float sintheta = sqrtf(1.0f - costheta*costheta);
			float theta = acos(costheta);
			float diff = theta - kt_cos[i - N_pol*N_mt*N_mt_max]; 
            if (sintheta < 1.e-6){
				if (diff < 0){
					diff *= 2.0f*K_kt*1000000;
				}
				else{
					diff *= -2.0f*K_kt*1000000;
				}
			}
			else{
				diff *= (-2.0f*K_kt*1000000)*powf(sintheta, -1);
			}
			float c1 = diff*r12inv;
			float c2 = diff*r32inv;
    
			float3 f1, f2, f3;
			f1.x = c1*(dr12.x*(r12inv*costheta) - dr32.x*r32inv);
			f1.y = c1*(dr12.y*(r12inv*costheta) - dr32.y*r32inv);
			f1.z = c1*(dr12.z*(r12inv*costheta) - dr32.z*r32inv);
			f2 = f1;
			f3.x = c2*(dr32.x*(r32inv*costheta) - dr12.x*r12inv);
			f3.y = c2*(dr32.y*(r32inv*costheta) - dr12.y*r12inv);
			f3.z = c2*(dr32.z*(r32inv*costheta) - dr12.z*r12inv);
			f2.x += f3.x;
			f2.y += f3.y;
			f2.z += f3.z;

			f2.x = -f2.x;
			f2.y = -f2.y;
			f2.z = -f2.z;

            atomicAdd(&f[i].x, f1.x);
			atomicAdd(&f[i].y, f1.y);		
			atomicAdd(&f[i].z, f1.z);
        }
        else    if (type[i] == RIGHT_NDC){
            int j, k, ks;

            for (ks = 1; ks < N_kt; ks += 2){
                if (i < N_pol*N_mt*N_mt_max + (ks + 1)*kn && i >= N_pol*N_mt*N_mt_max + ks*kn){
                    k = N_pol*N_mt*N_mt_max + ks*kn - 1 - chrom_bn;
                    j = N_pol*N_mt*N_mt_max + (ks + 1)*kn - 1 - chrom_bn;
                }
            }

			float3 r1 = r[i];
			float3 r2 = r[j];
			float3 r3 = r[k];
            
			float3 dr12, dr32;
			dr12.x = r1.x - r2.x;
			dr12.y = r1.y - r2.y;
			dr12.z = r1.z - r2.z;

			dr32.x = r3.x - r2.x;
			dr32.y = r3.y - r2.y;
			dr32.z = r3.z - r2.z;

			float r12inv = 1.0f*powf(dr12.x*dr12.x + dr12.y*dr12.y + dr12.z*dr12.z, -0.5);
			float r32inv = 1.0f*powf(dr32.x*dr32.x + dr32.y*dr32.y + dr32.z*dr32.z, -0.5);
			float costheta = (dr12.x*dr32.x + dr12.y*dr32.y + dr12.z*dr32.z)*r12inv*r32inv;
			if (costheta > 1.0f){
				costheta = 1.0f;
			}
			else	if (costheta < -1.0f){
				costheta = -1.0f;
			}
			float sintheta = sqrtf(1.0f - costheta*costheta);
			float theta = acos(costheta);
			float diff = theta - kt_cos[i - N_pol*N_mt*N_mt_max];
			if (sintheta < 1.e-6){
				if (diff < 0){
					diff *= 2.0f*K_kt*100000;
				}
				else{
					diff *= -2.0f*K_kt*100000;
				}
			}
			else{
				diff *= (-2.0f*K_kt*100000)*powf(sintheta, -1);
			}
			float c1 = diff*r12inv;
			float c2 = diff*r32inv;

			float3 f1, f2, f3;
			f1.x = c1*(dr12.x*(r12inv*costheta) - dr32.x*r32inv);
			f1.y = c1*(dr12.y*(r12inv*costheta) - dr32.y*r32inv);
			f1.z = c1*(dr12.z*(r12inv*costheta) - dr32.z*r32inv);
			f2 = f1;
			f3.x = c2*(dr32.x*(r32inv*costheta) - dr12.x*r12inv);
			f3.y = c2*(dr32.y*(r32inv*costheta) - dr12.y*r12inv);
			f3.z = c2*(dr32.z*(r32inv*costheta) - dr12.z*r12inv);
			f2.x += f3.x;
			f2.y += f3.y;
			f2.z += f3.z;

			f2.x = -f2.x;
			f2.y = -f2.y;
			f2.z = -f2.z;

            atomicAdd(&f[i].x, f1.x);
			atomicAdd(&f[i].y, f1.y);		
			atomicAdd(&f[i].z, f1.z);
        }
        else    if (type[i] == SHELL_RIGHT){
            int j, k;
            for (int ks = 1; ks < N_kt; ks += 2){
                if (i < N_pol*N_mt*N_mt_max + (ks + 1)*kn && i >= N_pol*N_mt*N_mt_max + ks*kn){
                    k = N_pol*N_mt*N_mt_max + ks*kn - 1 - chrom_bn;
                    j = N_pol*N_mt*N_mt_max + (ks + 1)*kn - 1 - chrom_bn;
                }
            }
			float3 r1 = r[i];
			float3 r2 = r[j];
			float3 r3 = r[k];

			float3 dr12, dr32;
			dr12.x = r1.x - r2.x;
			dr12.y = r1.y - r2.y;
			dr12.z = r1.z - r2.z;

			dr32.x = r3.x - r2.x;
			dr32.y = r3.y - r2.y;
			dr32.z = r3.z - r2.z;

			float r12inv = 1.0f*powf(dr12.x*dr12.x + dr12.y*dr12.y + dr12.z*dr12.z, -0.5);
			float r32inv = 1.0f*powf(dr32.x*dr32.x + dr32.y*dr32.y + dr32.z*dr32.z, -0.5);
			float costheta = (dr12.x*dr32.x + dr12.y*dr32.y + dr12.z*dr32.z)*r12inv*r32inv;
			if (costheta > 1.0f){
				costheta = 1.0f;
			}
			else	if (costheta < -1.0f){
				costheta = -1.0f;
			}
			float sintheta = sqrtf(1.0f - costheta*costheta);
			float theta = acos(costheta);
			float diff = theta - kt_cos[i - N_pol*N_mt*N_mt_max];
			if (sintheta < 1.e-6){
				if (diff < 0){
					diff *= 2.0f*K_kt*1000000;
				}
				else{
					diff *= -2.0f*K_kt*1000000;
				}
			}
			else{
				diff *= (-2.0f*K_kt*1000000)*powf(sintheta, -1);
			}
			float c1 = diff*r12inv;
			float c2 = diff*r32inv;

			float3 f1, f2, f3;
			f1.x = c1*(dr12.x*(r12inv*costheta) - dr32.x*r32inv);
			f1.y = c1*(dr12.y*(r12inv*costheta) - dr32.y*r32inv);
			f1.z = c1*(dr12.z*(r12inv*costheta) - dr32.z*r32inv);
			f2 = f1;
			f3.x = c2*(dr32.x*(r32inv*costheta) - dr12.x*r12inv);
			f3.y = c2*(dr32.y*(r32inv*costheta) - dr12.y*r12inv);
			f3.z = c2*(dr32.z*(r32inv*costheta) - dr12.z*r12inv);
			f2.x += f3.x;
			f2.y += f3.y;
			f2.z += f3.z;

			f2.x = -f2.x;
			f2.y = -f2.y;
			f2.z = -f2.z;

            atomicAdd(&f[i].x, f1.x);
			atomicAdd(&f[i].y, f1.y);		
			atomicAdd(&f[i].z, f1.z);
        }
    //	__syncthreads();
	}	
}	
