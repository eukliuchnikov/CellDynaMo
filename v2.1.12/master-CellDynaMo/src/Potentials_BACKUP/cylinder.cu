#include "hip/hip_runtime.h"
#include "cylinder.cuh"		

__global__ void cyl_cyl(float3* r, float3* f, int N, int* type, Param* d_parameters){
    int N_pol = d_parameters->pole_n;
    int N_mt = d_parameters->mt_n;
    int N_mt_max = d_parameters->max_mt_n;
    float e_lj = d_parameters->e_rep/0.23;
    float mt_rad = d_parameters->mt_r;
	int i = blockIdx.x*blockDim.x + threadIdx.x;	
	if (i < N_mt*N_mt_max){
        if (type[i] == MT_REG || type[i] == MT_END){
		    float3 ri1 = r[i];  //a0
		    float3 rj1 = r[i + 1];  //a1
            
            float3 intersect1, intersect2;
            float min_dist;
            float dx, dy, dz;

            float3 p_vector1, p1_norm;
            p_vector1.x = rj1.x - ri1.x;  //A
            p_vector1.y = rj1.y - ri1.y;
            p_vector1.z = rj1.z - ri1.z;

            float dp1 = sqrtf(p_vector1.x*p_vector1.x + p_vector1.y*p_vector1.y + p_vector1.z+ p_vector1.z); //magA
            p1_norm.x = p_vector1.x/dp1; //_A
            p1_norm.y = p_vector1.y/dp1;
            p1_norm.z = p_vector1.z/dp1;

            for (int j = N_mt*N_mt_max; j < N_pol*N_mt*N_mt_max; j++){
                if ((type[j] == MT_REG || type[j] == MT_END)){
		            float3 ri2 = r[j]; //b0
		            float3 rj2 = r[j + 1];  //b1

                    float3 p_vector2, p2_norm;
                    p_vector2.x = rj2.x - ri2.x; //B
                    p_vector2.y = rj2.y - ri2.y;
                    p_vector2.z = rj2.z - ri2.z;

                    float dp2 = sqrtf(p_vector2.x*p_vector2.x + p_vector2.y*p_vector2.y + p_vector2.z+ p_vector2.z); //magB
                    p2_norm.x = p_vector2.x/dp2; //_B
                    p2_norm.y = p_vector2.y/dp2;
                    p2_norm.z = p_vector2.z/dp2;

                    float3 cross;
                    cross.x = p1_norm.y*p2_norm.z - p2_norm.y*p1_norm.z; //cross
                    cross.y = p1_norm.z*p2_norm.x - p2_norm.z*p1_norm.x;
                    cross.z = p1_norm.x*p2_norm.y - p2_norm.x*p1_norm.y;

                    float d_cross = sqrtf(cross.x*cross.x + cross.y*cross.y + cross.z*cross.z); //denom
                    d_cross = d_cross*d_cross;
                    if (d_cross == 0){
                        float d0 = p1_norm.x*(ri2.x - ri1.x) + p1_norm.y*(ri2.y - ri1.y) + p1_norm.z*(ri2.z - ri1.z);
                        float d1 = p1_norm.x*(rj2.x - ri1.x) + p1_norm.y*(rj2.y - ri1.y) + p1_norm.z*(rj2.z - ri1.z);
                        if (d0 <= 0 && 0 >= d1){
                            if (sqrtf(d0*d0) < sqrtf(d1*d1)){
                                intersect1 = ri1;
                                intersect2 = ri2;
                                dx = intersect2.x - intersect1.x;
                                dy = intersect2.y - intersect1.y;
                                dz = intersect2.z - intersect1.z;
                                min_dist = sqrtf(dx*dx + dy*dy + dz*dz);
                            }
                            else    {
                                intersect1 = ri1;
                                intersect2 = rj2;
                                
                                dx = intersect2.x - intersect1.x;
                                dy = intersect2.y - intersect1.y;
                                dz = intersect2.z - intersect1.z;
                                min_dist = sqrtf(dx*dx + dy*dy + dz*dz);
                            }
                        }
                        else if (d0 >= dp1 && dp1 <= d1){
                            if (sqrtf(d0*d0) < sqrtf(d1*d1)){
                                intersect1 = rj1;
                                intersect2 = ri2;
                                dx = intersect2.x - intersect1.x;
                                dy = intersect2.y - intersect1.y;
                                dz = intersect2.z - intersect1.z;
                                min_dist = sqrtf(dx*dx + dy*dy + dz*dz);
                            }
                            else    {
                                intersect1 = rj1;
                                intersect2 = rj2;
                                dx = intersect2.x - intersect1.x;
                                dy = intersect2.y - intersect1.y;
                                dz = intersect2.z - intersect1.z;
                                min_dist = sqrtf(dx*dx + dy*dy + dz*dz);
                            }
                        }
                    }
                    else {
                        float3 t;
                        t.x = ri2.x - ri1.x;
                        t.y = ri2.y - ri1.y;
                        t.z = ri2.z - ri1.z;

                        float det0 = t.x*(p2_norm.y*cross.z - cross.y*p2_norm.z) + t.y*(p2_norm.z*cross.x - cross.z*p2_norm.x) + t.z*(p2_norm.x*cross.y - cross.x*p2_norm.y);
                        float det1 = t.x*(p1_norm.y*cross.z - cross.y*p1_norm.z) + t.y*(p1_norm.z*cross.x - cross.z*p1_norm.x) + t.z*(p1_norm.x*cross.y - cross.x*p1_norm.y);
                        float t0 = det0/d_cross;
                        float t1 = det1/d_cross;

                        float3 p1, p2;
                        p1.x = ri1.x + t0*p1_norm.x;
                        p1.y = ri1.y + t0*p1_norm.y;
                        p1.z = ri1.z + t0*p1_norm.z;

                        p2.x = ri2.x + t1*p2_norm.x;
                        p2.y = ri2.y + t1*p2_norm.y;
                        p2.z = ri2.z + t1*p2_norm.z;

                        if (t0 < 0){
                            p1.x = ri1.x;
                            p1.y = ri1.y;
                            p1.z = ri1.z;
                        }
                        else if (t0 > dp1){
                            p1.x = ri2.x;
                            p1.y = ri2.y;
                            p1.z = ri2.z;
                        }
                        if (t1 < 0){
                            p2.x = rj1.x;
                            p2.y = rj1.y;
                            p2.z = rj1.z;
                        }
                        else if (t1 > dp2){
                            p2.x = rj2.x;
                            p2.y = rj2.y;
                            p2.z = rj2.z;
                        }
                        float dot;
                        if (t0 < 0 || t0 > dp1){
                            dot = p2_norm.x*(p1.x - rj1.x) + p2_norm.y*(p1.y - rj1.y) + p2_norm.z*(p1.z - rj1.z);
                            if (dot < 0){
                                dot = 0.0;
                            }
                            else if (dot > dp2){
                                dot = dp2;
                            }
                            p2.x = rj1.x + dot*p2_norm.x;
                            p2.y = rj1.y + dot*p2_norm.y;
                            p2.z = rj1.z + dot*p2_norm.z;
                        }

                        if (t1 < 0 || t1 > dp2){
                            dot = p1_norm.x*(p2.x - ri1.x) + p1_norm.y*(p2.y - ri1.y) + p1_norm.z*(p2.z - ri1.z);
                            if (dot < 0){
                                dot = 0.0;
                            }
                            else if (dot > dp1){
                                dot = dp1;
                            }
                            p1.x = ri1.x + dot*p1_norm.x;
                            p1.y = ri1.y + dot*p1_norm.y;
                            p1.z = ri1.z + dot*p1_norm.z;
                        }

                        intersect1 = p1;
                        intersect2 = p2;
                        dx = p2.x - p1.x;
                        dy = p2.y - p1.y;
                        dz = p2.z - p1.z;
                        min_dist = sqrtf(dx*dx + dy*dy + dz*dz);
                    }

                    if (min_dist < 2.0*mt_rad){
                        float3 d_a;
                        d_a.x = intersect1.x - ri1.x;
                        d_a.y = intersect1.y - ri1.y;
                        d_a.z = intersect1.z - ri1.z;
                        float alpha = sqrtf(d_a.x*d_a.x + d_a.y* d_a.y + d_a.z*d_a.z)/dp1; 

                        float3 d_b;
                        d_b.x = intersect2.x - ri2.x;
                        d_b.y = intersect2.y - ri2.y;
                        d_b.z = intersect2.z - ri2.z; 
                        float betha = sqrtf(d_b.x*d_b.x + d_b.y* d_b.y + d_b.z*d_b.z)/dp2;

                        float df = -6*e_lj*powf((2.0*mt_rad/min_dist), 12)*powf(min_dist, -1)*powf(min_dist, -1);
                        atomicAdd(&f[i].x, df*(1 - alpha)*dx);
                        atomicAdd(&f[i].y, df*(1 - alpha)*dy);  		
                        atomicAdd(&f[i].z, df*(1 - alpha)*dz);

                        atomicAdd(&f[i + 1].x, df*alpha*dx);
                        atomicAdd(&f[i + 1].y, df*alpha*dy);  		
                        atomicAdd(&f[i + 1].z, df*alpha*dz);

                        atomicAdd(&f[j].x, -df*(1 - betha)*dx);
                        atomicAdd(&f[j].y, -df*(1 - betha)*dy);		
                        atomicAdd(&f[j].z, -df*(1 - betha)*dz); 

                        atomicAdd(&f[j + 1].x, -df*betha*dx);
                        atomicAdd(&f[j + 1].y, -df*betha*dy);  		
                        atomicAdd(&f[j + 1].z, -df*betha*dz);
                    } 
                }
            }
        }
    }
}

__global__ void cyl_sphere(float3* r, float3* f, int N, int* type, int kn, Param* d_parameters){
    int N_pol = d_parameters->pole_n;
    int N_mt = d_parameters->mt_n;
    int N_kt = d_parameters->kin_n;
    int N_mt_max = d_parameters->max_mt_n;    
    float e_lj = d_parameters->e_rep/0.23;
    float mt_rad = d_parameters->mt_r;
    float kt_rad = d_parameters->kt_r;
    int chrom_cond = d_parameters->chrom;
    int x_num = d_parameters->x_num;
    int y_num = d_parameters->y_num;
    int chrom_bn = chrom_cond*d_parameters->chrom_num;
	int i = blockIdx.x*blockDim.x + threadIdx.x;	
	if (i < 2*N_pol*N_mt*N_mt_max){
        if (type[i] == MT_REG || type[i] == MT_END || type[i] == LEFT_MINUS || type[i] == RIGHT_MINUS){
            printf("CYL: %d\t%d\n", i, N_pol);
		    float3 ri1 = r[i];  //a0
		    float3 rj1 = r[i + 1];  //a1
            
            float3 intersect;
            float min_dist;
            float dx, dy, dz;

            float3 p_vector1, p1_norm;
            p_vector1.x = rj1.x - ri1.x;  //A
            p_vector1.y = rj1.y - ri1.y;
            p_vector1.z = rj1.z - ri1.z;

            float dp1 = sqrtf(p_vector1.x*p_vector1.x + p_vector1.y*p_vector1.y + p_vector1.z+ p_vector1.z); //magA
            p1_norm.x = p_vector1.x/dp1; //_A
            p1_norm.y = p_vector1.y/dp1;
            p1_norm.z = p_vector1.z/dp1;
            
            for (int h = 0; h < N_kt; h++){
		        int s = N_pol*N_mt*N_mt_max + (h + 1)*kn - 1 - chrom_bn;
                float3 rs = r[s];

                float3 p_vectorS, pS_Scale;
                p_vectorS.x = rs.x - ri1.x;  //A
                p_vectorS.y = rs.y - ri1.y;
                p_vectorS.z = rs.z - ri1.z;

                pS_Scale.x = p_vectorS.x/dp1;
                pS_Scale.y = p_vectorS.y/dp1;
                pS_Scale.z = p_vectorS.z/dp1;
                float dpS = sqrtf(p_vectorS.x*p_vectorS.x + p_vectorS.y*p_vectorS.y + p_vectorS.z+ p_vectorS.z); //magA

                float t = p1_norm.x*pS_Scale.x + p1_norm.y*pS_Scale.y + p1_norm.z*pS_Scale.z;
                if (t < 0.0){
                    t = 0.0;
                }
                if (t > 1.0){
                    t = 1.0;
                }
                intersect.x = p_vector1.x*t;
                intersect.y = p_vector1.y*t;
                intersect.z = p_vector1.z*t;
                dx = intersect.x - p_vectorS.x;
                dy = intersect.y - p_vectorS.y;
                dz = intersect.z - p_vectorS.z;
                min_dist = sqrtf(dx*dx + dy*dy + dz*dz);
               
                if (min_dist < mt_rad + kt_rad){
                    float3 d_a;
                    d_a.x = intersect.x - ri1.x;
                    d_a.y = intersect.y - ri1.y;
                    d_a.z = intersect.z - ri1.z;
                    float alpha = sqrtf(d_a.x*d_a.x + d_a.y* d_a.y + d_a.z*d_a.z)/dp1; 

                   // float df = -6*e_lj*powf(((mt_rad + kt_rad)/min_dist), 12)*powf(min_dist, -1)*powf(min_dist, -1);
                    float df = -6*10.0*powf(((mt_rad + kt_rad)/min_dist), 12)*powf(min_dist, -1)*powf(min_dist, -1)/10000;
                    atomicAdd(&f[i].x, -df*(1 - alpha)*dx);
                    atomicAdd(&f[i].y, -df*(1 - alpha)*dy);  		
                    atomicAdd(&f[i].z, -df*(1 - alpha)*dz);

                    atomicAdd(&f[i + 1].x, -df*alpha*dx);
                    atomicAdd(&f[i + 1].y, -df*alpha*dy);  		
                    atomicAdd(&f[i + 1].z, -df*alpha*dz);

                    atomicAdd(&f[s].x, df*dx);
                    atomicAdd(&f[s].y, df*dy);		
                    atomicAdd(&f[s].z, df*dz); 
                }
            }
            printf("check: %d\t%d", chrom_bn, N_kt);
            for (int p = 0; p < N_kt; p++){
                for (int ch = 1; ch <= chrom_bn; ch++){
                    int s = N_pol*N_mt*N_mt_max + (p + 1)*kn - ch;
                    printf("SAS:%d\n", s);
                    float3 rs = r[s];

                    float3 p_vectorS, pS_Scale;
                    p_vectorS.x = rs.x - ri1.x;  //A
                    p_vectorS.y = rs.y - ri1.y;
                    p_vectorS.z = rs.z - ri1.z;

                    pS_Scale.x = p_vectorS.x/dp1;
                    pS_Scale.y = p_vectorS.y/dp1;
                    pS_Scale.z = p_vectorS.z/dp1;
                    float dpS = sqrtf(p_vectorS.x*p_vectorS.x + p_vectorS.y*p_vectorS.y + p_vectorS.z+ p_vectorS.z); //magA

                    float t = p1_norm.x*pS_Scale.x + p1_norm.y*pS_Scale.y + p1_norm.z*pS_Scale.z;
                    if (t < 0.0){
                        t = 0.0;
                    }
                    if (t > 1.0){
                        t = 1.0;
                    }
                    intersect.x = p_vector1.x*t;
                    intersect.y = p_vector1.y*t;
                    intersect.z = p_vector1.z*t;
                    dx = intersect.x - p_vectorS.x;
                    dy = intersect.y - p_vectorS.y;
                    dz = intersect.z - p_vectorS.z;
                    min_dist = sqrtf(dx*dx + dy*dy + dz*dz);
                   
                    if (min_dist < mt_rad + kt_rad){
                        printf("CHROM\n");
                        float3 d_a;
                        d_a.x = intersect.x - ri1.x;
                        d_a.y = intersect.y - ri1.y;
                        d_a.z = intersect.z - ri1.z;
                        float alpha = sqrtf(d_a.x*d_a.x + d_a.y* d_a.y + d_a.z*d_a.z)/dp1; 

                        //float df = -6*e_lj*powf(((mt_rad + kt_rad)/min_dist), 12)*powf(min_dist, -1)*powf(min_dist, -1);
                        float df = -6*10.0*powf(((mt_rad + kt_rad)/min_dist), 12)*powf(min_dist, -1)*powf(min_dist, -1)/10000;
                        atomicAdd(&f[i].x, -df*(1 - alpha)*dx);
                        atomicAdd(&f[i].y, -df*(1 - alpha)*dy);  		
                        atomicAdd(&f[i].z, -df*(1 - alpha)*dz);

                        atomicAdd(&f[i + 1].x, -df*alpha*dx);
                        atomicAdd(&f[i + 1].y, -df*alpha*dy);  		
                        atomicAdd(&f[i + 1].z, -df*alpha*dz);

                        atomicAdd(&f[s].x, df*dx);
                        atomicAdd(&f[s].y, df*dy);		
                        atomicAdd(&f[s].z, df*dz); 
                    }
                }
                for (int sh = x_num*y_num; sh < kn; sh++){
                    int s = N_pol*N_mt*N_mt_max + p*kn + sh;
                    //printf("SHELL: %d\n", s);
                    
                    float3 rs = r[s];

                    float3 p_vectorS, pS_Scale;
                    p_vectorS.x = rs.x - ri1.x;  //A
                    p_vectorS.y = rs.y - ri1.y;
                    p_vectorS.z = rs.z - ri1.z;

                    pS_Scale.x = p_vectorS.x/dp1;
                    pS_Scale.y = p_vectorS.y/dp1;
                    pS_Scale.z = p_vectorS.z/dp1;
                    float dpS = sqrtf(p_vectorS.x*p_vectorS.x + p_vectorS.y*p_vectorS.y + p_vectorS.z+ p_vectorS.z); //magA

                    float t = p1_norm.x*pS_Scale.x + p1_norm.y*pS_Scale.y + p1_norm.z*pS_Scale.z;
                    if (t < 0.0){
                        t = 0.0;
                    }
                    if (t > 1.0){
                        t = 1.0;
                    }
                    intersect.x = p_vector1.x*t;
                    intersect.y = p_vector1.y*t;
                    intersect.z = p_vector1.z*t;
                    dx = intersect.x - p_vectorS.x;
                    dy = intersect.y - p_vectorS.y;
                    dz = intersect.z - p_vectorS.z;
                    min_dist = sqrtf(dx*dx + dy*dy + dz*dz);
                   
                    if (min_dist < mt_rad + kt_rad){
                        float3 d_a;
                        d_a.x = intersect.x - ri1.x;
                        d_a.y = intersect.y - ri1.y;
                        d_a.z = intersect.z - ri1.z;
                        float alpha = sqrtf(d_a.x*d_a.x + d_a.y* d_a.y + d_a.z*d_a.z)/dp1; 

                        //float df = -6*e_lj*powf(((mt_rad + kt_rad)/min_dist), 12)*powf(min_dist, -1)*powf(min_dist, -1);
                        float df = -6*10.0*powf(((mt_rad + kt_rad)/min_dist), 12)*powf(min_dist, -1)*powf(min_dist, -1)/10000;
                        atomicAdd(&f[i].x, -df*(1 - alpha)*dx);
                        atomicAdd(&f[i].y, -df*(1 - alpha)*dy);  		
                        atomicAdd(&f[i].z, -df*(1 - alpha)*dz);

                        atomicAdd(&f[i + 1].x, -df*alpha*dx);
                        atomicAdd(&f[i + 1].y, -df*alpha*dy);  		
                        atomicAdd(&f[i + 1].z, -df*alpha*dz);

                        /*atomicAdd(&f[s].x, df*dx);
                        atomicAdd(&f[s].y, df*dy);		
                        atomicAdd(&f[s].z, df*dz); */
                    }
                }
            }      
        }
    }                    
}

/*__global__ void end_force(float3* r, float3* f, int N){

	int i = blockIdx.x*blockDim.x + threadIdx.x;	
	if (i == 0){        
		atomicAdd(&f[i].y, 50.0);		
	}
	if (i == 1){        
		atomicAdd(&f[i].y, 50.0);		
	}
}*/
