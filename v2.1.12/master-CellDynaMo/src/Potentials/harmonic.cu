#include "hip/hip_runtime.h"
#include "harmonic.cuh"		

__global__ void computeHarmonic(float3* r, float3* f, int N, int* type, int kn, float rk, int* harmonicKinCount, int* harmonicKin, float* harmonicKinRadii, float* kt_radius, Param* d_parameters, float* length, int* connect1, int* connect2, float* len1, float* len2, float* f_out){
    int N_pol = d_parameters->pole_n;
    int N_mt = d_parameters->mt_n;
    int N_kt = d_parameters->kin_n;
    int N_mt_max = d_parameters->max_mt_n;
    float K_kt = d_parameters->k_kt;
    float K_mt = d_parameters->k_mt;
    int MpK = d_parameters->mHkPm;
    int chrom_cond = d_parameters->chrom;
    int chrom_bn = chrom_cond*d_parameters->chrom_num;
    int x_num = d_parameters->x_num;
    int y_num = d_parameters->y_num;
    float kt_rad = d_parameters->kt_r;
    float mt_rad = d_parameters->mt_r;
	int i;	
	i = blockIdx.x*blockDim.x + threadIdx.x;	
	if (i < N){
        if (type[i] == LIG_MT_ACTIVE){
            //printf("%d\t%d\t%d\n", i, type[i], LIG_MT_ACTIVE);
            float r0 = 50.0;
   			float3 ri = r[i];
			float3 rj = r[i + 1];

			float dx = rj.x - ri.x;
			float dy = rj.y - ri.y;
			float dz = rj.z - ri.z;
	        
			float dr = sqrtf(dx*dx + dy*dy + dz*dz);
            //printf("%d\t%d: %f\t%f\n", i, i - 1, dr, r0);
            if (dr > r0){
                float df = 10*K_mt*(dr - r0)*powf(dr, -1);
                f_out[i] = df;

	            atomicAdd(&f[i].x, df*dx);
	            atomicAdd(&f[i].y, df*dy);		
	            atomicAdd(&f[i].z, df*dz);
                
		        atomicAdd(&f[i + 1].x, -df*dx);
		        atomicAdd(&f[i + 1].y, -df*dy);		
		        atomicAdd(&f[i + 1].z, -df*dz);
            }
            int m1 = connect1[i];
            
            float3 r_m1 = r[m1];
            float dr1 = len1[i];

		    float dx1 = r_m1.x - ri.x;
		    float dy1 = r_m1.y - ri.y;
		    float dz1 = r_m1.z - ri.z;

            float dr_m1 = sqrtf(dx1*dx1 + dy1*dy1 + dz1*dz1);
            //printf("1: %d\t%d\t: %f\t%f\n", i, m1, dr_m1, dr1);
            //printf("%d\t%d: %f\t%f\n", i, i - 1, dr, r0);
            //if (dr_m1 > 2*mt_rad){
            float df1 = 100*K_mt*(dr_m1 - dr1)*powf(dr_m1, -1);
            if (df1 > 100){
                df1 = 100.0;
            }
            if (df1 < -100){
                df1 = -100.0;
            }

            atomicAdd(&f[i].x, df1*dx1);
            atomicAdd(&f[i].y, df1*dy1);		
            atomicAdd(&f[i].z, df1*dz1);
            
            atomicAdd(&f[m1].x, -df1*dx1);
            atomicAdd(&f[m1].y, -df1*dy1);		
            atomicAdd(&f[m1].z, -df1*dz1);

            int m2 = connect2[i];            
            float3 r_m2 = r[m2];
            float dr2 = len2[i];

            float dx2 = r_m2.x - ri.x;
		    float dy2 = r_m2.y - ri.y;
		    float dz2 = r_m2.z - ri.z;

            float dr_m2 = sqrtf(dx2*dx2 + dy2*dy2 + dz2*dz2);
            //printf("2: %d\t%d\t: %f\t%f\n", i, m2, dr_m2, dr2);
            //printf("%d\t%d: %f\t%f\n", i, i - 1, dr, r0);
            //if (dr_m2 > 2*mt_rad){
            float df2 = 100*K_mt*(dr_m2 - dr2)*powf(dr_m2, -1);
            if (df2 > 100){
                df2 = 100.0;
            }
            if (df2 < -100){
                df2 = -100.0;
            }

            atomicAdd(&f[i].x, df2*dx2);
            atomicAdd(&f[i].y, df2*dy2);		
            atomicAdd(&f[i].z, df2*dz2);
            
            atomicAdd(&f[m2].x, -df2*dx2);
            atomicAdd(&f[m2].y, -df2*dy2);		
            atomicAdd(&f[m2].z, -df2*dz2);
            //}

            float3 ri1 = r[connect1[i]];  //a0
		    float3 rj1 = r[connect2[i]];  //a1
            
            float3 intersect;
            float min_dist;
            float dxc, dyc, dzc;

            float3 p_vector1;
            p_vector1.x = rj1.x - ri1.x;  //AB
            p_vector1.y = rj1.y - ri1.y;
            p_vector1.z = rj1.z - ri1.z;

            float dp1 = sqrtf(p_vector1.x*p_vector1.x + p_vector1.y*p_vector1.y + p_vector1.z+ p_vector1.z); //magA

            float3 p_vectorA;
            p_vectorA.x = ri.x - ri1.x;  //AP
            p_vectorA.y = ri.y - ri1.y;
            p_vectorA.z = ri.z - ri1.z;

            float3 p_vectorB;
            p_vectorB.x = ri.x - rj1.x;  //BP
            p_vectorB.y = ri.y - rj1.y;
            p_vectorB.z = ri.z - rj1.z;

            float AB_BP_dot = (p_vector1.x*p_vectorB.x + p_vector1.y*p_vectorB.y + p_vector1.z*p_vectorB.z);
            float AB_AP_dot = (p_vector1.x*p_vectorA.x + p_vector1.y*p_vectorA.y + p_vector1.z*p_vectorA.z);

            if (AB_BP_dot > 0){
                dxc = ri.x - rj1.x;
                dyc = ri.y - rj1.y;
                dzc = ri.z - rj1.z;
                min_dist = sqrtf(dxc*dxc + dyc*dyc + dzc*dzc);
                intersect = rj1;
            }
            else    if (AB_AP_dot < 0){
                dxc = ri.x - ri1.x;
                dyc = ri.y - ri1.y;
                dzc = ri.z - ri1.z;
                min_dist = sqrtf(dxc*dxc + dyc*dyc + dzc*dzc);
                intersect = ri1;
            }
            else    {
                float3 cross;
                cross.x = p_vectorA.y*p_vectorB.z - p_vectorB.y*p_vectorA.z; //cross
                cross.y = p_vectorA.z*p_vectorB.x - p_vectorB.z*p_vectorA.x;
                cross.z = p_vectorA.x*p_vectorB.y - p_vectorB.x*p_vectorA.y;

                float d_cross = sqrtf(cross.x*cross.x + cross.y*cross.y + cross.z*cross.z); //denom
                min_dist = d_cross/dp1;

                float t = AB_AP_dot/powf(dp1, 2);
                intersect.x = ri1.x + p_vector1.x*t;
                intersect.y = ri1.y + p_vector1.y*t;
                intersect.z = ri1.z + p_vector1.z*t;
                dxc = ri.x - intersect.x;
                dyc = ri.y - intersect.y;
                dzc = ri.z - intersect.z;
            }

            float3 d_a;
            d_a.x = intersect.x - ri1.x;
            d_a.y = intersect.y - ri1.y;
            d_a.z = intersect.z - ri1.z;
            float alpha = sqrtf(d_a.x*d_a.x + d_a.y* d_a.y + d_a.z*d_a.z)/dp1; 
            //printf("MT: %d\t%d\t%d\t%f\t%f\n", i, connect1[i], connect2[i], min_dist, mt_rad);
            float dfc = 10*K_mt*(min_dist - mt_rad)*powf(min_dist, -1);
            if (dfc > 100.0){
                dfc = 100.0;
            } 
            if (dfc < -100.0){
                dfc = -100.0;
            } 

            /*atomicAdd(&f[connect1[i]].x, dfc*(1 - alpha)*dxc);
            atomicAdd(&f[connect1[i]].y, dfc*(1 - alpha)*dyc);  		
            atomicAdd(&f[connect1[i]].z, dfc*(1 - alpha)*dzc);

            atomicAdd(&f[connect2[i]].x, dfc*alpha*dxc);
            atomicAdd(&f[connect2[i]].y, dfc*alpha*dyc);  		
            atomicAdd(&f[connect2[i]].z, dfc*alpha*dzc);

            atomicAdd(&f[i].x, -dfc*dxc);
            atomicAdd(&f[i].y, -dfc*dyc);		
            atomicAdd(&f[i].z, -dfc*dzc);*/ 
        }

        if (type[i] == LIG_CH_ACTIVE){
   			float3 ri = r[i];
			float3 rj = r[i + 1];
            int m1 = connect1[i];
           
            float3 r_m1 = r[m1];
            float dr1 = len1[i];

		    float dx1 = r_m1.x - ri.x;
		    float dy1 = r_m1.y - ri.y;
		    float dz1 = r_m1.z - ri.z;

            float dr_m1 = sqrtf(dx1*dx1 + dy1*dy1 + dz1*dz1);

            float df1 = 10*K_mt*(dr_m1 - dr1)*powf(dr_m1, -1);

            atomicAdd(&f[i].x, df1*dx1);
            atomicAdd(&f[i].y, df1*dy1);		
            atomicAdd(&f[i].z, df1*dz1);
            
            atomicAdd(&f[m1].x, -df1*dx1);
            atomicAdd(&f[m1].y, -df1*dy1);		
            atomicAdd(&f[m1].z, -df1*dz1);

            int m2 = connect2[i];            
            float3 r_m2 = r[m2];
            float dr2 = len2[i];
            
            float dx2 = r_m2.x - ri.x;
		    float dy2 = r_m2.y - ri.y;
		    float dz2 = r_m2.z - ri.z;

            float dr_m2 = sqrtf(dx2*dx2 + dy2*dy2 + dz2*dz2);
            //printf("%d\t%d: %f\t%f\n", i, i - 1, dr, r0);

            float df2 = 10*K_mt*(dr_m2 - dr2)*powf(dr_m2, -1);

            atomicAdd(&f[i].x, df2*dx2);
            atomicAdd(&f[i].y, df2*dy2);		
            atomicAdd(&f[i].z, df2*dz2);
            
            atomicAdd(&f[m2].x, -df2*dx2);
            atomicAdd(&f[m2].y, -df2*dy2);		
            atomicAdd(&f[m2].z, -df2*dz2);

            float3 ri1 = r[connect1[i]];  //a0
		    float3 rj1 = r[connect2[i]];  //a1
            
            float3 intersect;
            float min_dist;
            float dxc, dyc, dzc;

            float3 p_vector1;
            p_vector1.x = rj1.x - ri1.x;  //AB
            p_vector1.y = rj1.y - ri1.y;
            p_vector1.z = rj1.z - ri1.z;

            float dp1 = sqrtf(p_vector1.x*p_vector1.x + p_vector1.y*p_vector1.y + p_vector1.z+ p_vector1.z); //magA

            float3 p_vectorA;
            p_vectorA.x = ri.x - ri1.x;  //AP
            p_vectorA.y = ri.y - ri1.y;
            p_vectorA.z = ri.z - ri1.z;

            float3 p_vectorB;
            p_vectorB.x = ri.x - rj1.x;  //BP
            p_vectorB.y = ri.y - rj1.y;
            p_vectorB.z = ri.z - rj1.z;

            float AB_BP_dot = (p_vector1.x*p_vectorB.x + p_vector1.y*p_vectorB.y + p_vector1.z*p_vectorB.z);
            float AB_AP_dot = (p_vector1.x*p_vectorA.x + p_vector1.y*p_vectorA.y + p_vector1.z*p_vectorA.z);

            if (AB_BP_dot > 0){
                dxc = ri.x - rj1.x;
                dyc = ri.y - rj1.y;
                dzc = ri.z - rj1.z;
                min_dist = sqrtf(dxc*dxc + dyc*dyc + dzc*dzc);
                intersect = rj1;
            }
            else    if (AB_AP_dot < 0){
                dxc = ri.x - ri1.x;
                dyc = ri.y - ri1.y;
                dzc = ri.z - ri1.z;
                min_dist = sqrtf(dxc*dxc + dyc*dyc + dzc*dzc);
                intersect = ri1;
            }
            else    {
                float3 cross;
                cross.x = p_vectorA.y*p_vectorB.z - p_vectorB.y*p_vectorA.z; //cross
                cross.y = p_vectorA.z*p_vectorB.x - p_vectorB.z*p_vectorA.x;
                cross.z = p_vectorA.x*p_vectorB.y - p_vectorB.x*p_vectorA.y;

                float d_cross = sqrtf(cross.x*cross.x + cross.y*cross.y + cross.z*cross.z); //denom
                min_dist = d_cross/dp1;

                float t = AB_AP_dot/powf(dp1, 2);
                intersect.x = ri1.x + p_vector1.x*t;
                intersect.y = ri1.y + p_vector1.y*t;
                intersect.z = ri1.z + p_vector1.z*t;
                dxc = ri.x - intersect.x;
                dyc = ri.y - intersect.y;
                dzc = ri.z - intersect.z;
            }

            float3 d_a;
            d_a.x = intersect.x - ri1.x;
            d_a.y = intersect.y - ri1.y;
            d_a.z = intersect.z - ri1.z;
            float alpha = sqrtf(d_a.x*d_a.x + d_a.y* d_a.y + d_a.z*d_a.z)/dp1; 
            //printf("CH: %d\t%d\t%d\t%f\t%f\n", i, connect1[i], connect2[i], min_dist, kt_rad);
            float dfc = 10*K_mt*(min_dist - kt_rad)*powf(min_dist, -1);
            if (dfc > 100.0){
                dfc = 100.0;
            } 
            if (dfc < -100.0){
                dfc = -100.0;
            } 

            atomicAdd(&f[connect1[i]].x, dfc*(1 - alpha)*dxc);
            atomicAdd(&f[connect1[i]].y, dfc*(1 - alpha)*dyc);  		
            atomicAdd(&f[connect1[i]].z, dfc*(1 - alpha)*dzc);

            atomicAdd(&f[connect2[i]].x, dfc*alpha*dxc);
            atomicAdd(&f[connect2[i]].y, dfc*alpha*dyc);  		
            atomicAdd(&f[connect2[i]].z, dfc*alpha*dzc);

            atomicAdd(&f[i].x, -dfc*dxc);
            atomicAdd(&f[i].y, -dfc*dyc);		
            atomicAdd(&f[i].z, -dfc*dzc);
        }
		if (type[i] == MT_REG || type[i] == PLUS_DET || type[i] == PLUS_ATT || type[i] == PLUS_DET_INVALID){
            float r0 = length[i]/(N_mt_max - 1);
            if (r0 > 12*mt_rad){
       			float3 ri = r[i - 1];
			    float3 rj = r[i];

			    float dx = rj.x - ri.x;
			    float dy = rj.y - ri.y;
			    float dz = rj.z - ri.z;
	            
			    float dr = sqrtf(dx*dx + dy*dy + dz*dz);
                //printf("%d\t%d: %f\t%f\n", i, i - 1, dr, r0);
                float df = 100*K_mt*(dr - r0)*powf(dr, -1);

		        atomicAdd(&f[i - 1].x, df*dx);
		        atomicAdd(&f[i - 1].y, df*dy);		
		        atomicAdd(&f[i - 1].z, df*dz);
                
			    atomicAdd(&f[i].x, -df*dx);
			    atomicAdd(&f[i].y, -df*dy);		
			    atomicAdd(&f[i].z, -df*dz);
            }		
        }
        if (type[i] == LEFT_MINUS){
            float r0 = length[i]/(N_mt_max - 1);
            if (r0 > 12*mt_rad){
       			float3 ri = r[N - 2];
			    float3 rj = r[i];

			    float dx = rj.x - ri.x;
			    float dy = rj.y - ri.y;
			    float dz = rj.z - ri.z;
	        
			    float dr = sqrtf(dx*dx + dy*dy + dz*dz);
                float df = 100*K_mt*(dr - r0)*powf(dr, -1);

		        atomicAdd(&f[N - 2].x, df*dx);
		        atomicAdd(&f[N - 2].y, df*dy);		
		        atomicAdd(&f[N - 2].z, df*dz);
                
			    atomicAdd(&f[i].x, -df*dx);
			    atomicAdd(&f[i].y, -df*dy);		
			    atomicAdd(&f[i].z, -df*dz);
            }
		}
        if (type[i] == RIGHT_MINUS){
            float r0 = length[i]/(N_mt_max - 1);
            if (r0 > 12*mt_rad){
       			float3 ri = r[N - 1];
			    float3 rj = r[i];

			    float dx = rj.x - ri.x;
			    float dy = rj.y - ri.y;
			    float dz = rj.z - ri.z;
	        
			    float dr = sqrtf(dx*dx + dy*dy + dz*dz);
                float df = 100*K_mt*(dr - r0)*powf(dr, -1);

		        atomicAdd(&f[N - 1].x, df*dx);
		        atomicAdd(&f[N - 1].y, df*dy);		
		        atomicAdd(&f[N - 1].z, df*dz);
                
			    atomicAdd(&f[i].x, -df*dx);
			    atomicAdd(&f[i].y, -df*dy);		
			    atomicAdd(&f[i].z, -df*dz);
            }
		}
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//**************************************************************************************************************************************************************//
//between centres of kinetochores
//**************************************************************************************************************************************************************//
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		if (type[i] == LEFT_KT){
			float3 ri = r[i];
			int j = i + kn;
			float3 rj = r[j];

			float dx = rj.x - ri.x;
			float dy = rj.y - ri.y;
			float dz = rj.z - ri.z;

			float dr = sqrtf(dx*dx + dy*dy + dz*dz);
			float df = 5.0*K_kt*(dr - rk)*powf(dr, -1);

			atomicAdd(&f[i].x, df*dx);
			atomicAdd(&f[i].y, df*dy);		
			atomicAdd(&f[i].z, df*dz);

			atomicAdd(&f[j].x, -df*dx);
			atomicAdd(&f[j].y, -df*dy);		
			atomicAdd(&f[j].z, -df*dz);
            
            if (chrom_bn != 0){
                for (int chrom = 1; chrom < chrom_bn + 1; chrom++){
                    float3 rch1 = r[i + chrom];
			        int c1 = i + kn + chrom;
			        float3 rch2 = r[c1];

			        float dx1 = rch2.x - rch1.x;
			        float dy1 = rch2.y - rch1.y;
			        float dz1 = rch2.z - rch1.z;

			        float dr1 = sqrtf(dx1*dx1 + dy1*dy1 + dz1*dz1);
			        float df1 = 0.001*K_kt*(dr1 - rk)*powf(dr1, -1);
                    
                    atomicAdd(&f[i + chrom].x, df1*dx1);
			        atomicAdd(&f[i + chrom].y, df1*dy1);		
			        atomicAdd(&f[i + chrom].z, df1*dz1);

			        atomicAdd(&f[c1].x, -df1*dx1);
			        atomicAdd(&f[c1].y, -df1*dy1);		
			        atomicAdd(&f[c1].z, -df1*dz1);
                }

                for (int chrom = 1; chrom < chrom_bn; chrom += chrom_bn/2){
                    int k = i + chrom;
			        float3 rk = r[k];

			        float dxk = rk.x - ri.x;
			        float dyk = rk.y - ri.y;
			        float dzk = rk.z - ri.z;


			        float drk = sqrtf(dxk*dxk + dyk*dyk + dzk*dzk);
			        float dfk = 200*K_kt*(drk - kt_radius[k - N_pol*N_mt*N_mt_max])*powf(drk, -1);
                    //printf("LKT: %d\t%d\t%f\t%f\n", i, k, drk, kt_radius[k - N_pol*N_mt*N_mt_max]);

			        atomicAdd(&f[i].x, dfk*dxk);
			        atomicAdd(&f[i].y, dfk*dyk);		
			        atomicAdd(&f[i].z, dfk*dzk);

			        atomicAdd(&f[k].x, -dfk*dxk);
			        atomicAdd(&f[k].y, -dfk*dyk);		
			        atomicAdd(&f[k].z, -dfk*dzk);
                }
            }
		}
        if (type[i] == RIGHT_KT && chrom_bn != 0){
            float3 ri = r[i];
            
            for (int chrom = 1; chrom < chrom_bn; chrom += chrom_bn/2){
                int k = i + chrom;
			    float3 rk = r[k];

			    float dxk = rk.x - ri.x;
			    float dyk = rk.y - ri.y;
			    float dzk = rk.z - ri.z;

			    float drk = sqrtf(dxk*dxk + dyk*dyk + dzk*dzk);
			    float dfk = 200*K_kt*(drk - kt_radius[k - N_pol*N_mt*N_mt_max])*powf(drk, -1);
               // printf("RKT: %d\t%d\t%f\t%f\n", i, k, drk, kt_radius[k - N_pol*N_mt*N_mt_max]);

			    atomicAdd(&f[i].x, dfk*dxk);
			    atomicAdd(&f[i].y, dfk*dyk);		
			    atomicAdd(&f[i].z, dfk*dzk);

			    atomicAdd(&f[k].x, -dfk*dxk);
			    atomicAdd(&f[k].y, -dfk*dyk);		
			    atomicAdd(&f[k].z, -dfk*dzk);
            }
		}
        if (type[i] == CHROM){

            float3 ri = r[i];
            int ks;
            for (ks = 0; ks < N_kt; ks ++){            
                if (i < N_pol*N_mt*N_mt_max + (ks + 1)*kn - chrom_bn/2 && i > N_pol*N_mt*N_mt_max + (ks + 1)*kn - chrom_bn || i < N_pol*N_mt*N_mt_max + (ks + 1)*kn && i > N_pol*N_mt*N_mt_max + (ks + 1)*kn - chrom_bn/2){
                    int k = i - 1;
			        float3 rk = r[k];
			        float dxk = rk.x - ri.x;
			        float dyk = rk.y - ri.y;
			        float dzk = rk.z - ri.z;

			        float drk = sqrtf(dxk*dxk + dyk*dyk + dzk*dzk);
			        float dfk = 200*K_kt*(drk - kt_radius[i - N_pol*N_mt*N_mt_max])*powf(drk, -1);
                   // printf("CHROM: %d\t%d\t%f\t%f\n", i, k, drk, kt_radius[k - N_pol*N_mt*N_mt_max]);
                    //printf("%d\t%d\t%f\t%f\n", i, k, drk, kt_radius[i - N_pol*N_mt*N_mt_max]);
			        atomicAdd(&f[i].x, dfk*dxk);
			        atomicAdd(&f[i].y, dfk*dyk);		
			        atomicAdd(&f[i].z, dfk*dzk);

			        atomicAdd(&f[k].x, -dfk*dxk);
			        atomicAdd(&f[k].y, -dfk*dyk);		
			        atomicAdd(&f[k].z, -dfk*dzk);
                }
            }
        }
        if (type[i] == LEFT_NDC){
			float3 ri = r[i];
            int j, mult, ks;
            for (ks = 0; ks < N_kt; ks += 2){
                if (i < N_pol*N_mt*N_mt_max + (ks + 1)*kn && i >= N_pol*N_mt*N_mt_max + ks*kn){
                    j = N_pol*N_mt*N_mt_max + (ks + 1)*kn - 1 - chrom_bn;
                    mult = ks;
                }
            }
            float3 rj = r[j];

			float dx = rj.x - ri.x;
			float dy = rj.y - ri.y;
			float dz = rj.z - ri.z;

			float dr = sqrtf(dx*dx + dy*dy + dz*dz);
			float df = K_kt*(dr - kt_radius[i - N_pol*N_mt*N_mt_max])*powf(dr, -1);

			atomicAdd(&f[i].x, df*dx);
			atomicAdd(&f[i].y, df*dy);		
			atomicAdd(&f[i].z, df*dz);
            
            int l;          
            for (l = 0; l < harmonicKinCount[i - N_pol*N_mt*N_mt_max]; l++){
				float3 rl = r[N_pol*N_mt*N_mt_max + kn*mult + harmonicKin[(i - N_pol*N_mt*N_mt_max)*MpK + l]];

                float dxl = rl.x - ri.x;
				float dyl = rl.y - ri.y;
				float dzl = rl.z - ri.z;	
				float drl = sqrtf(dxl*dxl + dyl*dyl + dzl*dzl);
               
				float dfl = K_kt*(drl - harmonicKinRadii[(i - N_pol*N_mt*N_mt_max)*MpK + l])*powf(drl, -1)/100;
                             
                atomicAdd(&f[i].x, dfl*dxl);
			    atomicAdd(&f[i].y, dfl*dyl);		
			    atomicAdd(&f[i].z, dfl*dzl);

                atomicAdd(&f[N_pol*N_mt*N_mt_max + kn*mult + harmonicKin[(i - N_pol*N_mt*N_mt_max)*MpK + l]].x, -dfl*dxl);
			    atomicAdd(&f[N_pol*N_mt*N_mt_max + kn*mult + harmonicKin[(i - N_pol*N_mt*N_mt_max)*MpK + l]].y, -dfl*dyl);	
			    atomicAdd(&f[N_pol*N_mt*N_mt_max + kn*mult + harmonicKin[(i - N_pol*N_mt*N_mt_max)*MpK + l]].z, -dfl*dzl);
            }
        }
        /*if (type[i] == SHELL_LEFT){
			float3 ri = r[i];
            int j, mult, ks;
            for (ks = 0; ks < N_kt; ks += 2){
                if (i < N_pol*N_mt*N_mt_max + (ks + 1)*kn && i >= N_pol*N_mt*N_mt_max + ks*kn){
                    j = N_pol*N_mt*N_mt_max + (ks + 1)*kn - 1 - chrom_bn;
                    mult = ks;
                }
            }

            float3 rj = r[j];

			float dx = rj.x - ri.x;
			float dy = rj.y - ri.y;
			float dz = rj.z - ri.z;

			float dr = sqrtf(dx*dx + dy*dy + dz*dz);
			float df = K_kt*(dr - kt_radius[i - N_pol*N_mt*N_mt_max])*powf(dr, -1);

			atomicAdd(&f[i].x, df*dx);
			atomicAdd(&f[i].y, df*dy);		
			atomicAdd(&f[i].z, df*dz);
            
            int l;          
            for (l = 0; l < harmonicKinCount[i - N_pol*N_mt*N_mt_max]; l++){
				float3 rl = r[N_pol*N_mt*N_mt_max + kn*mult + harmonicKin[(i - N_pol*N_mt*N_mt_max)*MpK + l]];

                float dxl = rl.x - ri.x;
				float dyl = rl.y - ri.y;
				float dzl = rl.z - ri.z;	
				float drl = sqrtf(dxl*dxl + dyl*dyl + dzl*dzl);
               
				float dfl = K_kt*(drl - harmonicKinRadii[(i - N_pol*N_mt*N_mt_max)*MpK + l])*powf(drl, -1);
                             
                atomicAdd(&f[i].x, dfl*dxl);
			    atomicAdd(&f[i].y, dfl*dyl);		
			    atomicAdd(&f[i].z, dfl*dzl);

                atomicAdd(&f[N_pol*N_mt*N_mt_max + kn*mult + harmonicKin[(i - N_pol*N_mt*N_mt_max)*MpK + l]].x, -dfl*dxl);
			    atomicAdd(&f[N_pol*N_mt*N_mt_max + kn*mult + harmonicKin[(i - N_pol*N_mt*N_mt_max)*MpK + l]].y, -dfl*dyl);	
			    atomicAdd(&f[N_pol*N_mt*N_mt_max + kn*mult + harmonicKin[(i - N_pol*N_mt*N_mt_max)*MpK + l]].z, -dfl*dzl);
            }
        }*/
        if (type[i] == RIGHT_NDC){
			float3 ri = r[i];
            int j, mult, ks;
            for (ks = 1; ks < N_kt; ks += 2){
                if (i < N_pol*N_mt*N_mt_max + (ks + 1)*kn && i >= N_pol*N_mt*N_mt_max + ks*kn){
                    j = N_pol*N_mt*N_mt_max + (ks + 1)*kn - 1 - chrom_bn;
                    mult = ks;
                }
            }
            float3 rj = r[j];

			float dx = rj.x - ri.x;
			float dy = rj.y - ri.y;
			float dz = rj.z - ri.z;

			float dr = sqrtf(dx*dx + dy*dy + dz*dz);
			float df = K_kt*(dr - kt_radius[i - N_pol*N_mt*N_mt_max])*powf(dr, -1);
            //printf("%d\t%f\t%f\t%f\n", i, dr, kt_radius[i - N_pol*N_mt*N_mt_max], df);

			atomicAdd(&f[i].x, df*dx);
			atomicAdd(&f[i].y, df*dy);		
			atomicAdd(&f[i].z, df*dz);

            int l;
            for (l = 0; l < harmonicKinCount[i - N_pol*N_mt*N_mt_max]; l++){
			    float3 rl = r[N_pol*N_mt*N_mt_max + kn*mult + harmonicKin[(i - N_pol*N_mt*N_mt_max)*MpK + l]];
                float dxl = rl.x - ri.x;
			    float dyl = rl.y - ri.y;
			    float dzl = rl.z - ri.z;	
			    float drl = sqrtf(dxl*dxl + dyl*dyl + dzl*dzl);

			    float dfl = K_kt*(drl - harmonicKinRadii[(i - N_pol*N_mt*N_mt_max)*MpK + l])*powf(drl, -1)/100;
                //printf("%d\t%f\t%f\n", i, drl, harmonicKinRadii[(i - N_pol*N_mt*N_mt_max)*MpK + l]);                
                atomicAdd(&f[i].x, dfl*dxl);
		        atomicAdd(&f[i].y, dfl*dyl);		
		        atomicAdd(&f[i].z, dfl*dzl);

                atomicAdd(&f[N_pol*N_mt*N_mt_max + kn*mult + harmonicKin[(i - N_pol*N_mt*N_mt_max)*MpK + l]].x, -dfl*dxl);
		        atomicAdd(&f[N_pol*N_mt*N_mt_max + kn*mult + harmonicKin[(i - N_pol*N_mt*N_mt_max)*MpK + l]].y, -dfl*dyl);		
		        atomicAdd(&f[N_pol*N_mt*N_mt_max + kn*mult + harmonicKin[(i - N_pol*N_mt*N_mt_max)*MpK + l]].z, -dfl*dzl);
            }
        }
        /*if (type[i] == SHELL_RIGHT){
			float3 ri = r[i];
            int j, mult, ks;
            for (ks = 1; ks < N_kt; ks += 2){
                if (i < N_pol*N_mt*N_mt_max + (ks + 1)*kn && i >= N_pol*N_mt*N_mt_max + ks*kn){
                    j = N_pol*N_mt*N_mt_max + (ks + 1)*kn - 1 - chrom_bn;
                    mult = ks;
                }
            }
            float3 rj = r[j];

			float dx = rj.x - ri.x;
			float dy = rj.y - ri.y;
			float dz = rj.z - ri.z;

			float dr = sqrtf(dx*dx + dy*dy + dz*dz);
			float df = K_kt*(dr - kt_radius[i - N_pol*N_mt*N_mt_max])*powf(dr, -1);
            //printf("%d\t%f\t%f\t%f\n", i, dr, kt_radius[i - N_pol*N_mt*N_mt_max], df);

			atomicAdd(&f[i].x, df*dx);
			atomicAdd(&f[i].y, df*dy);		
			atomicAdd(&f[i].z, df*dz);

            int l;
            for (l = 0; l < harmonicKinCount[i - N_pol*N_mt*N_mt_max]; l++){
			    float3 rl = r[N_pol*N_mt*N_mt_max + kn*mult + harmonicKin[(i - N_pol*N_mt*N_mt_max)*MpK + l]];
                float dxl = rl.x - ri.x;
			    float dyl = rl.y - ri.y;
			    float dzl = rl.z - ri.z;	
			    float drl = sqrtf(dxl*dxl + dyl*dyl + dzl*dzl);

			    float dfl = K_kt*(drl - harmonicKinRadii[(i - N_pol*N_mt*N_mt_max)*MpK + l])*powf(drl, -1);
                //printf("%d\t%f\t%f\n", i, drl, harmonicKinRadii[(i - N_pol*N_mt*N_mt_max)*MpK + l]);                
                atomicAdd(&f[i].x, dfl*dxl);
		        atomicAdd(&f[i].y, dfl*dyl);		
		        atomicAdd(&f[i].z, dfl*dzl);

                atomicAdd(&f[N_pol*N_mt*N_mt_max + kn*mult + harmonicKin[(i - N_pol*N_mt*N_mt_max)*MpK + l]].x, -dfl*dxl);
		        atomicAdd(&f[N_pol*N_mt*N_mt_max + kn*mult + harmonicKin[(i - N_pol*N_mt*N_mt_max)*MpK + l]].y, -dfl*dyl);		
		        atomicAdd(&f[N_pol*N_mt*N_mt_max + kn*mult + harmonicKin[(i - N_pol*N_mt*N_mt_max)*MpK + l]].z, -dfl*dzl);
            }
        }*/
    //	__syncthreads();
	}
}
