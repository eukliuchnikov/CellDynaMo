#include "hip/hip_runtime.h"
#include "kt_ex_vol.cuh"		

__global__ void excl_vol_kt(float3* r, float3* f, int N, int* type, int kn, Param* d_parameters){
	int i, j;
    int N_pol = d_parameters->pole_n;
    int N_mt = d_parameters->mt_n;
    int N_kt = d_parameters->kin_n;
    int N_mt_max = d_parameters->max_mt_n;
    float mt_rad = d_parameters->mt_r;
    float kt_rad = d_parameters->kt_r;
    float e_lj = d_parameters->e_rep/0.23;
    int chrom_cond = d_parameters->chrom;
    int chrom_bn = chrom_cond*d_parameters->chrom_num;
    int x_num = d_parameters->x_num;
    int y_num = d_parameters->y_num;
    //float e_mor = d_parameters->e_att/0.23;
	i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < N){
		if (type[i] == LEFT_KT){
			float3 ri = r[i];
            for ((j = N_pol*N_mt*N_mt_max + kn - 1); j < N; j += kn){
                if (i != j && j != (i + kn)){
                    float3 rj = r[j];
                    float dx = rj.x - ri.x;
	                float dy = rj.y - ri.y;
	                float dz = rj.z - ri.z;
                    float dr = sqrtf(dx*dx + dy*dy + dz*dz);
                    if (dr < 2*(kt_rad + mt_rad)){
                        float df = -6*powf((2*(mt_rad + kt_rad)/dr), 12)*powf(dr, -1)*powf(dr, -1)*e_lj;
                        atomicAdd(&f[i].x, df*dx);
	                    atomicAdd(&f[i].y, df*dy);  		
	                    atomicAdd(&f[i].z, df*dz);

	                    atomicAdd(&f[j].x, -df*dx);
	                    atomicAdd(&f[j].y, -df*dy);		
	                    atomicAdd(&f[j].z, -df*dz);
                    }
                }
            }
            int k = i + kn;
            float3 rk = r[k];
            float dx = rk.x - ri.x;
            float dy = rk.y - ri.y;
            float dz = rk.z - ri.z;
            float dr = sqrtf(dx*dx + dy*dy + dz*dz);
            if (dr < 2*(kt_rad)){
                float df = -6*powf((2*(kt_rad)/dr), 12)*powf(dr, -1)*powf(dr, -1)*e_lj;
                atomicAdd(&f[i].x, df*dx);
                atomicAdd(&f[i].y, df*dy);  		
                atomicAdd(&f[i].z, df*dz);

                atomicAdd(&f[k].x, -df*dx);
                atomicAdd(&f[k].y, -df*dy);		
                atomicAdd(&f[k].z, -df*dz);
            }
		}
        if (type[i] == RIGHT_KT){
			float3 ri = r[i];
            for ((j = N_pol*N_mt*N_mt_max + kn - 1); j < N; j += kn){
                if (i != j && j != (i - kn)){
                    float3 rj = r[j];
                    float dx = rj.x - ri.x;
	                float dy = rj.y - ri.y;
	                float dz = rj.z - ri.z;
                    float dr = sqrtf(dx*dx + dy*dy + dz*dz);
                    if (dr < 2*(kt_rad + mt_rad)){
                        float df = -6*powf((2*(mt_rad + kt_rad)/dr), 12)*powf(dr, -1)*powf(dr, -1)*e_lj;
                        atomicAdd(&f[i].x, df*dx);
	                    atomicAdd(&f[i].y, df*dy);  		
	                    atomicAdd(&f[i].z, df*dz);

	                    atomicAdd(&f[j].x, -df*dx);
	                    atomicAdd(&f[j].y, -df*dy);		
	                    atomicAdd(&f[j].z, -df*dz);
                    }
                }
            }
		}
    	if (type[i] == CHROM){
			float3 ri = r[i];
            int ks;
            int c_id = i - N_pol*N_mt*N_mt_max;
            int k_id = int(c_id/kn);
            for (ks = 0; ks < N_kt; ks++){ 
                for (int ch = 1; ch <= chrom_bn + 1; ch++){
                    int k = N_pol*N_mt*N_mt_max + (ks + 1)*kn - ch;
                    if (ks != k_id){
                        float3 rj = r[k];
                        float dx = rj.x - ri.x;
	                    float dy = rj.y - ri.y;
	                    float dz = rj.z - ri.z;
                        float dr = sqrtf(dx*dx + dy*dy + dz*dz);
                        if (dr < 2*kt_rad){
                            float df = -6*powf((2*kt_rad/dr), 12)*powf(dr, -1)*powf(dr, -1)*e_lj;
                            atomicAdd(&f[i].x, df*dx);
	                        atomicAdd(&f[i].y, df*dy);  		
	                        atomicAdd(&f[i].z, df*dz);

	                        atomicAdd(&f[k].x, -df*dx);
	                        atomicAdd(&f[k].y, -df*dy);		
	                        atomicAdd(&f[k].z, -df*dz);
                        }
                    }
                }
            }
            int ic = i - (kn - x_num*y_num) - 0.5*x_num*y_num;
             
            float3 rc = r[ic];
            float dx = rc.x - ri.x;
            float dy = rc.y - ri.y;
            float dz = rc.z - ri.z;
            float dr = sqrtf(dx*dx + dy*dy + dz*dz);
            if (dr < sqrtf(2.0)*kt_rad){
                float df = -6*powf((sqrtf(2.0)*kt_rad/dr), 12)*powf(dr, -1)*powf(dr, -1)*e_lj;
                atomicAdd(&f[i].x, df*dx);
                atomicAdd(&f[i].y, df*dy);  		
                atomicAdd(&f[i].z, df*dz);

                /*atomicAdd(&f[ic].x, -df*dx);
                atomicAdd(&f[ic].y, -df*dy);		
                atomicAdd(&f[ic].z, -df*dz);*/
            }
        }
    	//__syncthreads();
	}
}	

__global__ void chrom_ter(float3* r, float3* f, int N, int* type, int kn, Param* d_parameters){
	int i;
    float pol_dist = d_parameters->pole_dist;
    float pole_rad = d_parameters->pole_r;
    float kt_rad = d_parameters->kt_r;
    //float e_mor = d_parameters->e_att/0.23;
	i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < N){
		if (type[i] == LEFT_KT || type[i] == RIGHT_KT || type[i] == CHROM){
			float re2 = pow(r[i].x, 2)*pow(pol_dist - 1.0625*pole_rad - 1.5*kt_rad, -2) + pow(r[i].y, 2)*pow(pol_dist - 1.0625*pole_rad - 1.5*kt_rad, -2) + pow(r[i].z, 2)*pow(pol_dist - 1.0625*pole_rad - 1.5*kt_rad, -2); 
			if (re2 > 1.0){
                float dr = sqrtf(r[i].x*r[i].x + r[i].y*r[i].y + r[i].z*r[i].z);
				atomicAdd(&f[i].x, -100000*(re2 - 1.0)*r[i].x*powf(dr, -1));
                atomicAdd(&f[i].y, -100000*(re2 - 1.0)*r[i].y*powf(dr, -1));  		
                atomicAdd(&f[i].z, -100000*(re2 - 1.0)*r[i].z*powf(dr, -1));
			}
		}
	}
}	
__global__ void chrom_memb(float3* r, float3* f, int N, int* type, int kn, Param* d_parameters){
	int i;
    float kt_rad = d_parameters->kt_r;
    float mt_rad = d_parameters->mt_r;
    float a_rad = d_parameters->a;
    float b_rad = d_parameters->b;
    float c_rad = d_parameters->c;
    //float e_mor = d_parameters->e_att/0.23;
	i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < N){
		if (type[i] == LEFT_KT || type[i] == RIGHT_KT || type[i] == CHROM){
			float re2 = pow(r[i].x, 2)*pow(a_rad - kt_rad, -2) + pow(r[i].y, 2)*pow(b_rad - kt_rad, -2) + pow(r[i].z, 2)*pow(c_rad - kt_rad, -2); 
			if (re2 > 1.0){
                float dr = sqrtf(r[i].x*r[i].x + r[i].y*r[i].y + r[i].z*r[i].z);
				atomicAdd(&f[i].x, -100000*(re2 - 1.0)*r[i].x*powf(dr, -1));
                atomicAdd(&f[i].y, -100000*(re2 - 1.0)*r[i].y*powf(dr, -1));  		
                atomicAdd(&f[i].z, -100000*(re2 - 1.0)*r[i].z*powf(dr, -1));
			}
		}
        if (type[i] == MT_REG || type[i] == PLUS_DET || type[i] == PLUS_ATT || type[i] == PLUS_DET_INVALID){
			float re2 = pow(r[i].x, 2)*pow(a_rad - 2*mt_rad, -2) + pow(r[i].y, 2)*pow(b_rad - 2*mt_rad, -2) + pow(r[i].z, 2)*pow(c_rad - 2*mt_rad, -2); 
			if (re2 > 1.0){
                float dr = sqrtf(r[i].x*r[i].x + r[i].y*r[i].y + r[i].z*r[i].z);
				atomicAdd(&f[i].x, -100000*(re2 - 1.0)*r[i].x*powf(dr, -1));
                atomicAdd(&f[i].y, -100000*(re2 - 1.0)*r[i].y*powf(dr, -1));  		
                atomicAdd(&f[i].z, -100000*(re2 - 1.0)*r[i].z*powf(dr, -1));
			}
		}
	}
}		
