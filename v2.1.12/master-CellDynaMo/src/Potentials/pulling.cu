#include "hip/hip_runtime.h"
#include "pulling.cuh"		

__global__ void pulling(float3* r, float3* f, int N, int* type, int kn, long long int step, int* n_force, float* f_force, Param* d_parameters, int* connector, float* length, float* att_l, float* link_l){
	int i, j;
    int N_pol = d_parameters->pole_n;
    int N_mt = d_parameters->mt_n;
    int N_kt = d_parameters->kin_n;
    int N_mt_max = d_parameters->max_mt_n;
    float mt_rad = d_parameters->mt_r;
    float e_mor = d_parameters->e_att;
    int chrom_cond = d_parameters->chrom;
    int chrom_bn = chrom_cond*d_parameters->chrom_num;
    i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < N){
		if (type[i] == PLUS_ATT){
			float3 ri = r[i];
           
            float3 rp = r[i - 1];
            float dxp = rp.x - ri.x;
            float dyp = rp.y - ri.y;
            float dzp = rp.z - ri.z;

            float drp = sqrtf(dxp*dxp + dyp*dyp + dzp*dzp);
            float3 rpp = r[i - 2];
            float dxpp = rpp.x - rp.x;
            float dypp = rpp.y - rp.y;
            float dzpp = rpp.z - rp.z;

            float drpp = sqrtf(dxpp*dxpp + dypp*dypp + dzpp*dzpp);
            int pol;
            if (i < N_mt*N_mt_max){
                pol = N - 2;
            }
            else    if (i < N_pol*N_mt*N_mt_max){
                pol = N - 1;
            }
            if (drp + drpp > att_l[i] + mt_rad){
                length[i] = att_l[i];
                length[i - 1] = att_l[i];
                length[i - 2] = att_l[i];
                float dx0 = r[i - 2].x - r[pol].x;
			    float dy0 = r[i - 2].y - r[pol].y;
			    float dz0 = r[i - 2].z - r[pol].z;
			    float dr0 = sqrt(dx0*dx0 + dy0*dy0 + dz0*dz0);
                r[i].x = r[i - 1].x - (length[i]/2)*dxp/drp;
                r[i].y = r[i - 1].y - (length[i]/2)*dyp/drp;
                r[i].z = r[i - 1].z - (length[i]/2)*dyp/drp;

                ri = r[i];
            }
            if (connector[i] != 0){
                j = connector[i];
	            float3 rj = r[j];
	            float dx = rj.x - ri.x;
	            float dy = rj.y - ri.y;
	            float dz = rj.z - ri.z;

	            float dr = sqrtf(dx*dx + dy*dy + dz*dz);
              
                float3 r_pole1 = r[N - 2];
                float3 r_pole2 = r[N - 1];

                float dx1 = r_pole1.x - ri.x;
                float dy1 = r_pole1.y - ri.y;
                float dz1 = r_pole1.z - ri.z;

                float dx2 = r_pole2.x - ri.x;
                float dy2 = r_pole2.y - ri.y;
                float dz2 = r_pole2.z - ri.z;
    
                float dr1 = sqrtf(dx1*dx1 + dy1*dy1 + dz1*dz1);
                float dr2 = sqrtf(dx2*dx2 + dy2*dy2 + dz2*dz2);
                if (dr1 > 0.3*MIN_MT_LENGTH && dr2 > 0.3*MIN_MT_LENGTH){
                    float df = e_mor*(dr - link_l[i])*powf(dr, -1);
                    atomicAdd(&f[i].x, df*dx);
                    atomicAdd(&f[i].y, df*dy);		
                    atomicAdd(&f[i].z, df*dz);                   
                        
                    atomicAdd(&f[j].x, -df*dx);
	                atomicAdd(&f[j].y, -df*dy);		
	                atomicAdd(&f[j].z, -df*dz);
                    for (int ks = 0; ks < N_kt; ks++){
                        if (type[j] == LEFT_KT && j == N_pol*N_mt*N_mt_max + (ks + 1)*kn - 1 - chrom_bn  && step == 0){
                            atomicAdd(&n_force[ks*5], 1);
                            atomicAdd(&f_force[ks*5], df*1.66);
                            atomicAdd(&n_force[ks*5 + 1], 1);

                            /*atomicAdd(&n_force[ks*5], 1);
                            atomicAdd(&f_force[ks*5], df*1.66);
                            atomicAdd(&n_force[ks*5 + 4], 1);*/
                        }
                        else    if (type[j] == RIGHT_KT && j == N_pol*N_mt*N_mt_max + (ks + 1)*kn - 1 - chrom_bn  && step == 0){
                            atomicAdd(&n_force[ks*5], 1);
                            atomicAdd(&f_force[ks*5], df*1.66);
                            atomicAdd(&n_force[ks*5 + 1], 1);
                            
                            /*atomicAdd(&n_force[ks*5 + 1], 1);
                            atomicAdd(&f_force[ks*5 + 1], df*1.66);
                            atomicAdd(&n_force[ks*5 + 5], 1);*/
                        }
                    }
                }
            }
	    }
    	//__syncthreads();
	}
}		
